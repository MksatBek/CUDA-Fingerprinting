
#include "hip/hip_runtime.h"
#include ""
#include "MinutiaHelper.cuh"
#include "DescriptorBuilder.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>


int main()
{
	hipSetDevice(0);
	int i, j;
	int sizeOfMin = sizeof(Minutia);
	int sizeOfDesc = sizeof(Descriptor);
	int sizeOfInt = sizeof(int);
	int minPitch = MAX_DESC_SIZE * sizeOfMin;

	printf("1000x128 blocks with 128 threads each\n");
	printf("1000 lists of minutia database into base of lists of descriptors\n");
	for (int k = 0; k < 10; k++)
	{
		printf("\nrun num %d:\n", k);
		/////////work with input finger
		int fingerMinutiaNum;
		int *dev_fingerMinutiaNum;
		hipMalloc((void**)&dev_fingerMinutiaNum, sizeOfInt);
		char *fingerPath = "D:\\inputFinger.txt";

		Minutia *fingerMins = (Minutia*)malloc(minPitch);

		Minutia *dev_fingerMins;
		hipMalloc((void**)&dev_fingerMins, minPitch);

		fingerRead(fingerPath, fingerMins, &fingerMinutiaNum);
		hipMemcpy(dev_fingerMins, fingerMins, minPitch, hipMemcpyHostToDevice);
		hipMemcpy(dev_fingerMinutiaNum, &fingerMinutiaNum, sizeOfInt, hipMemcpyHostToDevice);

		Descriptor *dev_fingerDesc;
		hipMalloc((void**)&dev_fingerDesc, sizeOfDesc);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		buildDescriptors <<<dim3(1, MAX_DESC_SIZE), MAX_DESC_SIZE >>>(dev_fingerMins, 1, dev_fingerMinutiaNum, dev_fingerDesc, 1);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float et;
		hipEventElapsedTime(&et, start, stop);
		printf("for finger   %3.1fms \n", et);


		free(fingerMins);
		hipFree(dev_fingerMinutiaNum);
		hipFree(dev_fingerMins);
		hipFree(dev_fingerDesc);

		/////////end of work with input finger






		/////////work with fingers base
		int dbSize = 1000;
		int *dbMinutiaNum = (int*)malloc(dbSize*sizeOfInt);
		int *dev_dbMinutiaNum;
		hipMalloc((void**)&dev_dbMinutiaNum, dbSize*sizeOfInt);
		char *dbPath = "D:\\FingersBase";


		Minutia *dbMins = (Minutia*)malloc(dbSize * minPitch);

		Minutia *dev_dbMins;
			hipMalloc((void**)&dev_dbMins, dbSize * minPitch);

		fingersBaseRead(dbPath, dbSize, MAX_DESC_SIZE, dbMins, dbMinutiaNum); // done
		hipMemcpy(dev_dbMinutiaNum, dbMinutiaNum, dbSize * sizeOfInt, hipMemcpyHostToDevice);
		hipMemcpy(dev_dbMins, dbMins, minPitch*dbSize, hipMemcpyHostToDevice);

		Descriptor *dev_dbDesc;
		hipMalloc((void**)&dev_dbDesc, MAX_DESC_SIZE*dbSize*sizeOfDesc);

		hipEventRecord(start, 0);

		buildDescriptors <<<dim3(dbSize, MAX_DESC_SIZE), MAX_DESC_SIZE >>>(dev_dbMins, MAX_DESC_SIZE, dev_dbMinutiaNum, dev_dbDesc, dbSize);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&et, start, stop);
		printf("for base     %3.1fms\n", et);

		free(dbMinutiaNum);
		free(dbMins);
		hipFree(dev_dbMinutiaNum);
		hipFree(dev_dbMins);
		hipFree(dev_dbDesc);
		///////end of work with fingers base



	}
	printf("\n");



	/*
	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&prop, i);
		printf("name %s\n", prop.name);
		printf("overlap %d\n", prop.deviceOverlap);
		printf("multiproc %d\n", prop.multiProcessorCount);
		printf("threads %d\n", prop.maxThreadsPerBlock);
		printf("threads per dem %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("clockRate %d\n", prop.clockRate);
		printf("asyncEngines %d\n", prop.asyncEngineCount);
		printf("multyKernels %d\n", prop.concurrentKernels);
		printf("maxGridSize %d\n", prop.maxGridSize);
		
	}*/
}
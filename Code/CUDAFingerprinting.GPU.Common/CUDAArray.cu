#include "CUDAArray.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>


template<typename T> CUDAArray<T>::CUDAArray()
	{
	}

template<typename T> CUDAArray<T>::CUDAArray(const CUDAArray& arr)
	{
		cudaPtr = arr.cudaPtr;
		Height = arr.Height;
		Width = arr.Width;
		Stride = arr.Stride;
		deviceStride = arr.deviceStride;
	}

template<typename T> CUDAArray<T>::CUDAArray(T* cpuPtr, int width, int height)
	{
		Width = width;
		Height = height;
		hipError_t error = hipMallocPitch((void**)&cudaPtr, &Stride, Width*sizeof(T), Height);
		error = hipDeviceSynchronize();
		deviceStride = Stride / sizeof(T);
		error = hipMemcpy2D(cudaPtr, Stride, cpuPtr, Width*sizeof(T),
			Width*sizeof(T), Height, hipMemcpyHostToDevice);
		error = hipDeviceSynchronize();
		error = hipGetLastError();
	}

template<typename T> CUDAArray<T>::CUDAArray(int width, int height)
	{
		Width = width;
		Height = height;
		hipError_t error = hipMallocPitch((void**)&cudaPtr, &Stride, Width*sizeof(T), Height);
		error = hipDeviceSynchronize();
		deviceStride = Stride / sizeof(T);
	}

template<typename T> T* CUDAArray<T>::GetData()
	{
		T* arr = (T*)malloc(sizeof(T)*Width*Height);
		GetData(arr);
		return arr;
	}

template<typename T> void CUDAArray<T>::GetData(T* arr)
	{
		hipError_t error = hipMemcpy2D(arr, Width*sizeof(T), cudaPtr, Stride, Width*sizeof(T), Height, hipMemcpyDeviceToHost);
		error = hipDeviceSynchronize();
	}

template<typename T> __device__ T CUDAArray<T>::At(int row, int column)
	{
		return cudaPtr[row*deviceStride + column];
	}

template<typename T> __device__ void CUDAArray<T>::SetAt(int row, int column, T value)
	{
		cudaPtr[row*deviceStride + column] = value;
	}

template<typename T> void CUDAArray<T>::Dispose()
	{
		hipFree(cudaPtr);
	}

template<typename T> CUDAArray<T>::~CUDAArray()
	{

	}
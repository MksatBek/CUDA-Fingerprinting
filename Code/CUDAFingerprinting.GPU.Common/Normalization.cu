#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <math.h>
#include "CUDAArray.cuh"
float CalculateMean(CUDAArray<float> image)
{
	int height = image.Height;
	int width = image.Width;
	float mean = 0;
	for (int i = 0; i < width; i++)
	{
		for (int j = 0; j < height; j++)
		{
			mean += image.At(i, j) / (height * width);
		}
	}
	return mean;
}

float CalculateVariation(CUDAArray<float> image, float mean)
{
	int height = image.Height;
	int width = image.Width;
	float variation = 0;
	for (int i = 0; i < width; i++)
	{
		for (int j = 0; j < height; j++)
		{
			variation += pow((image.At(i, j) - mean), 2) / (height * width);
		}
	}
	return variation;
}
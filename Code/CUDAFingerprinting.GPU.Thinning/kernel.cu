//TEST FILE
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
//

#include "Thinning.cuh"

#include "ImageLoading.cuh"
#include "Utils.h"
//

//#define PROFILERENABLED

#ifdef PROFILERENABLED
#include "hip/hip_runtime_api.h"
#endif

int main()
{
	hipSetDevice(0);
	int width = 0;
	int height = 0;
	int* img = loadBmp("D:\\Ucheba\\Programming\\summerSchool\\Code\\Debug\\idealH.bmp", &width, &height);//test file from folder with executable file

#ifdef PROFILERENABLED
	hipProfilerStart();
#endif
	double** skeleton = Thin(intToDoubleArray(img, width, height), width, height);
#ifdef PROFILERENABLED
	hipProfilerStop();
#endif
	double** res = OverlapArrays(skeleton, intToDoubleArray(img, width, height), width, height);
	saveBmp("D:\\Ucheba\\Programming\\summerSchool\\Code\\Debug\\resultCUDA.bmp", doubleToIntArray(res, width, height), width, height);

	free(skeleton);
	free(res);
	system("D:\\Ucheba\\Programming\\summerSchool\\Code\\Debug\\resultCUDA.bmp");

	free(img);
	hipDeviceReset();
	return 0;
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ImageLoading.cuh"
#include "CUDAArray.cuh"
#include "Convolution.cuh"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <iostream>

using namespace std;

#define edge 50
#define pixEdge 150
#define defaultBlockSize 16

extern "C"
{
	__declspec( dllexport ) __global__ void cudaSegmentate (CUDAArray<float> value, int* matrix);
}

__global__ void SobelFilter (CUDAArray<float> source, CUDAArray<float> filterX, CUDAArray<float> filterY)
{
	int row = defaultRow ();
	int column = defaultColumn ();

	if (column < source.Width && row < source.Height && column > 0 && row > 0 )
	{
        float sumX = source.At(row - 1, column - 1) * filterX.At(0, 0) + source.At(row + 1, column - 1) * filterX.At(0, 2) +
                        source.At(row - 1, column) * filterX.At(1, 0) + source.At(row + 1, column) * filterX.At(1, 2) +
                        source.At(row - 1, column + 1) * filterX.At(2, 0) + source.At(row + 1, column + 1) * filterX.At(2, 2);
        
		float sumY = source.At(row - 1, column - 1) * filterY.At(0, 0) + source.At(row, column - 1) * filterY.At(0, 1) + source.At(row + 1, column - 1) * filterY.At(0, 2) +
            source.At(row - 1, column + 1) * filterY.At(2, 0) + source.At(row, column + 1) * filterY.At(2, 1) + source.At(row + 1, column + 1) * filterY.At(2, 2);
        
		float sqrtXY = sqrt(sumX * sumX + sumY * sumY);

        sqrtXY = sqrtXY > 255 ? 255 : sqrtXY;

		source.SetAt(row, column, sqrtXY);
    }
}

__global__ void cudaMatrix (CUDAArray<float> value, CUDAArray<int> matrix2D)
{
	int row = blockIdx.y * defaultBlockSize + threadIdx.y;
	int column = blockIdx.x * defaultBlockSize + threadIdx.x;

	int tX = threadIdx.x;
	int tY = threadIdx.y;

	__shared__ float buf[defaultBlockSize][defaultBlockSize];
	buf[tX][tY] = value.At (row, column);

	__syncthreads();
	if ( tX == 0 )
	{
		float sum = 0;
		for ( int i = 0; i < defaultBlockSize; ++i )
		{
			sum += buf[i][tY];
		}
		buf[0][tY] = sum;
	}

	__syncthreads();
	if ( tX == 0 && tY == 0 )
	{
		float sum = 0;
		for ( int i = 0; i < defaultBlockSize; ++i )
		{
			sum += buf[0][i];
		}
		buf[0][0] = sum;
	}
	__syncthreads();

	float val = buf[0][0] / ( defaultBlockSize * defaultBlockSize );

	if ( val >= edge )
	{
		if ( buf[tX][tY] < pixEdge )
		{
			matrix2D.SetAt (row, column, 1);
		}
		else
		{
			matrix2D.SetAt (row, column, 0);
		}
	}
	else
	{
		matrix2D.SetAt (row, column, 0);
	}
}

void Segmentate (CUDAArray<float> value, int* matrix)
{
	int width = value.Width;
	int height = value.Height;

	dim3 blockSize = dim3(defaultBlockSize, defaultBlockSize);
	dim3 gridSize = dim3(ceilMod(value.Width, defaultBlockSize), ceilMod(value.Height, defaultBlockSize));

	CUDAArray<int> matrix2D = CUDAArray<int>(matrix, width, height);
	cudaMatrix << < gridSize, blockSize >> >(value, matrix2D);

	matrix2D.GetData(matrix);

	ofstream f;
	f.open ("matrix.txt");
	for ( int i = 0; i < value.Width; ++i )
	{
		for ( int j = 0; j < value.Height; ++j )
		{
			f << matrix[i * value.Width + j] << ' ';
		}
		f << endl;
	}

	matrix2D.Dispose();
}

void BWPicture (int width, int height, int* matrix)
{
	int* newPic = (int*) malloc (sizeof (int)*width*height);
	int capacity = width * height;

	for ( int i = 0; i < capacity; ++i )
	{
		newPic[i] = matrix[i] * 255;
	}

	saveBmp ("newPic.bmp", newPic, width, height);

	free (newPic);
}

void MakingMatrix (float* fPic, int picWidth, int picHeight, int* matrix)
{
	CUDAArray<float> source = CUDAArray<float>(fPic, picWidth, picHeight);

	float filterXLinear[9] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
	float filterYLinear[9] = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };

	CUDAArray<float> filterX(filterXLinear, 3, 3);
	CUDAArray<float> filterY(filterYLinear, 3, 3);

	dim3 blockSize = dim3(defaultBlockSize, defaultBlockSize);
	dim3 gridSize = dim3(ceilMod(picWidth, defaultBlockSize), ceilMod(picHeight, defaultBlockSize));

	SobelFilter <<< gridSize, blockSize >>> (source, filterX, filterY);	
	
	//Saving image after Sobel Filter
	float* fSOPic = (float*)malloc(sizeof(float)*source.Width*source.Height);
	source.GetData (fSOPic);
	int* SOPic = (int*)malloc(sizeof(int)*source.Width*source.Height);
	for ( int i = 0; i < picWidth*picHeight; i++ )
	{
		SOPic[i] = (int) fSOPic[i];
	}
	saveBmp ("SOPic.bmp", SOPic, picWidth, picHeight);
	
	Segmentate (source, matrix);
	BWPicture (picWidth, picHeight, matrix);

	source.Dispose();
	filterX.Dispose();
	filterY.Dispose();

	free (fSOPic);
	free (SOPic);
}

int main()
{
	hipSetDevice (0);

	int picWidth, picHeight;
	int* pic = loadBmp ("..//1_1.bmp", &picWidth, &picHeight);
	float* fPic  = (float*) malloc (sizeof (float)*picWidth*picHeight);
	for ( int i = 0; i < picWidth * picHeight; i++ )
	{
		fPic[i] = (float) pic[i];
	}

	int *matrix = (int*) malloc (picWidth * picHeight * sizeof(int));
	// In this matrix 1 means light shade of gray, and 0 means dark shade of gray 

	MakingMatrix (fPic, picWidth, picHeight, matrix);

	free(pic);
	free (fPic);
	free (matrix);

	return 0;
}
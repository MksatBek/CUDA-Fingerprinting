#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#define _USE_MATH_DEFINES
#include <math.h>
#include <fstream>
#include "OrientationFieldRegularization.cuh"
using namespace std;
static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
int CeilMod(int x, int y){
	return (x > 1024 ? (x + y - 1) / y : x);
}
__global__ void NewFilter(float* filter, float sigma)
{
	float commonDenom = (float)2.0 * sigma * sigma;
	float denominator = (float)M_PI * commonDenom;
	int x = (blockDim.x >> 1) - blockIdx.x;
	int y = (blockDim.x >> 1) - threadIdx.x;
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	filter[tid] = (float)(exp((float)(-((float)(x*x + y*y)) / commonDenom)) / denominator);
}
__global__ void VectorField(float *Fx, float *Fy, float *O, int width)
{
	if (blockIdx.y*blockDim.x + threadIdx.x < width)
	{
		int tid = blockIdx.y*blockDim.x + threadIdx.x + blockIdx.x*width;
		Fx[tid] = cos(2 * O[tid]);
		Fy[tid] = sin(2 * O[tid]);
	}
}
__global__ void FilterAndLocalOrient(float* O, float *Fx, float *Fy, int width, int sizeFil, float* filter)
{
	if (blockIdx.y*blockDim.x + threadIdx.x < width)
	{
		int tid = blockDim.x*blockIdx.y + threadIdx.x + blockIdx.x*width;
		float resX = 0;
		float resY = 0;
		int i1 = blockIdx.x + (sizeFil >> 1);
		for (int u = 0; u < sizeFil; u++, i1--){
			int j1 = blockIdx.y*blockDim.x + threadIdx.x + (sizeFil >> 1);
			for (int v = 0; v < sizeFil; v++, j1--)
			{
				if ((i1 >= 0) && (i1 < gridDim.x) && (j1 >= 0) && (j1 < width))
				{
					int tidFil = u*sizeFil + v;
					resX += filter[tidFil] * Fx[i1 * width + j1];
					resY += filter[tidFil] * Fy[i1 * width + j1];
				}
			}
		}
		O[tid] = (float)0.5 *(atan(resY / resX));
		if (resX <= 0 && resY >= 0) O[tid] += M_PI_2;
		else if (resX <= 0 && resY <= 0) O[tid] -= M_PI_2;
	}
}
void OrientationRegularizationPixels(float *Out, float* O, int height, int width, int sizeFil)
{
	float *dev_Fy, *dev_Fx, *dev_O;
	int countThr = CeilMod(width, 4);
	float sigma = float(sizeFil - 1) / 6;
	if (sizeFil % 2 == 0)
		sizeFil -= 1;
	float *dev_filter;
	HandleError(hipMalloc((void**)&dev_filter, (sizeof(float)*sizeFil*sizeFil)), __FILE__, __LINE__);
	NewFilter << <sizeFil, sizeFil >> >(dev_filter, sigma);
	HandleError(hipMalloc((void**)&dev_Fx, (sizeof(float)*height*width)), __FILE__, __LINE__);
	HandleError(hipMalloc((void**)&dev_Fy, (sizeof(float)*height*width)), __FILE__, __LINE__);
	HandleError(hipMalloc((void**)&dev_O, (sizeof(float)*height*width)), __FILE__, __LINE__);
	HandleError(hipMemcpy(dev_O, O, (sizeof(float)*height*width), hipMemcpyHostToDevice), __FILE__, __LINE__);
	VectorField << < dim3(height, (width + countThr - 1) / countThr), countThr >> >(dev_Fx, dev_Fy, dev_O, width);
	FilterAndLocalOrient << < dim3(height, (width + countThr - 1) / countThr), countThr >> >(dev_O, dev_Fx, dev_Fy, width, sizeFil, dev_filter);
	hipFree(dev_Fx);
	hipFree(dev_Fy);
	HandleError(hipMemcpy(Out, dev_O, sizeof(float)*height*width, hipMemcpyDeviceToHost), __FILE__, __LINE__);

	hipFree(dev_O);
	hipFree(dev_filter);
}
float* OrientationRegularizationPixels(float* O, int height, int width, int sizeFil)
{
	float *dev_Fy, *dev_Fx, *dev_O;
	int countThr = CeilMod(width, 4);
	float sigma = float(sizeFil - 1) / 6;
	if (sizeFil % 2 == 0)
		sizeFil -= 1;
	float *dev_filter;
	HandleError(hipMalloc((void**)&dev_filter, (sizeof(float)*sizeFil*sizeFil)), __FILE__, __LINE__);
	NewFilter << <sizeFil, sizeFil >> >(dev_filter, sigma);
	HandleError(hipMalloc((void**)&dev_Fx, (sizeof(float)*height*width)), __FILE__, __LINE__);
	HandleError(hipMalloc((void**)&dev_Fy, (sizeof(float)*height*width)), __FILE__, __LINE__);
	HandleError(hipMalloc((void**)&dev_O, (sizeof(float)*height*width)), __FILE__, __LINE__);
	HandleError(hipMemcpy(dev_O, O, (sizeof(float)*height*width), hipMemcpyHostToDevice), __FILE__, __LINE__);
	VectorField << < dim3(height, (width + countThr - 1) / countThr), countThr >> >(dev_Fx, dev_Fy, dev_O, width);
	FilterAndLocalOrient << < dim3(height, (width + countThr - 1) / countThr), countThr >> >(dev_O, dev_Fx, dev_Fy, width, sizeFil, dev_filter);
	hipFree(dev_Fx);
	hipFree(dev_Fy);
	float *Out = (float*)malloc(height*width*sizeof(float));
	HandleError(hipMemcpy(Out, dev_O, sizeof(float)*height*width, hipMemcpyDeviceToHost), __FILE__, __LINE__);
	hipFree(dev_O);
	hipFree(dev_filter);
	return Out;
}
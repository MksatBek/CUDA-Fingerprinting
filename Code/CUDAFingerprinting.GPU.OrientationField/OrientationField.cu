#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""
#include "Convolution.cuh"
#include "constsmacros.h"
#include "imageLoading.cuh"
#include "CUDAArray.cuh"
#include "OrientationField.cuh"

// ----------- GPU ----------- //

__global__ void cudaSetOrientationInPixels(CUDAArray<float> orientation, CUDAArray<float> gradientX, CUDAArray<float> gradientY){
	int centerRow = defaultRow();
	int centerColumn = defaultColumn();

	const int size = 16;
	const int center = size / 2;
	const int upperLimit = center - 1;
	
	float product[size][size];
	float sqrdiff[size][size];

	for (int i = -center; i <= upperLimit; i++){
		for (int j = -center; j <= upperLimit; j++){
			if (i + centerRow < 0 || i + centerRow > gradientX.Height || j + centerColumn < 0 || j + centerColumn > gradientX.Width){		// ����� �� ������� ��������
				product[i + center][j + center] = 0;
				sqrdiff[i + center][j + center] = 0;
			}
			else{
				float GxValue = gradientX.At(i + centerRow, j + centerColumn);
				float GyValue = gradientY.At(i + centerRow, j + centerColumn);
				product[i + center][j + center] = GxValue * GyValue;						// ������������ ������������
				sqrdiff[i + center][j + center] = GxValue * GxValue - GyValue * GyValue;	// �������� ���������
			}
		}
	}
	__syncthreads();  // ���� ���� ��� ���� ������� ����������

	float numerator = 0;
	float denominator = 0;
	// ���������� ����
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++){
			numerator += product[i][j];
			denominator += sqrdiff[i][j];
		}
	}
	__syncthreads();

	// ���������� �������� ���� ����������
	if (denominator == 0){
		orientation.SetAt(centerRow, centerColumn, M_PI_2);
	}
	else{
		orientation.SetAt(centerRow, centerColumn, M_PI_2 + atan2(2 * numerator, denominator) / 2.0f);
		if (orientation.At(centerRow, centerColumn) > M_PI_2)
		{
			float index = orientation.At(centerRow, centerColumn) - M_PI;
			orientation.SetAt(centerRow, centerColumn, index);
		}
	}
}

__global__ void cudaSetOrientationInBlocks(CUDAArray<float> orientation, CUDAArray<float> gradientX, CUDAArray<float> gradientY){
	float numerator;
	float denominator;

	int column = defaultColumn();			// ���������� ������� -- ������� ������� �� ����� ��������
	int row = defaultRow();
	int threadColumn = threadIdx.x;			// ��������� ������� -- ������� ������� � ����� 
	int threadRow = threadIdx.y;
	float GyValue = gradientY.At(row, column);
	float GxValue = gradientX.At(row, column);

	const int defaultBlockSize = 16;		// ������ �����, �� �������� ��������� �����������

	// ���������� ��������� � �����������
	// ������� ����������� ��������������� �������� �������, ��������� �������� � shared ������
	__shared__ float product[defaultBlockSize][defaultBlockSize];
	__shared__ float sqrdiff[defaultBlockSize][defaultBlockSize];

	product[threadRow][threadColumn] = GxValue * GyValue; // �������� � ����� ������ ������������ ��������������� ��������� 
	sqrdiff[threadRow][threadColumn] = GxValue * GxValue - GyValue * GyValue; // �������� ���������
	__syncthreads();  // ���� ���� ��� ���� ������� ����������

	// ������ ����� �������������� �������� ������
	// ��������� �������� �����, ���������� ����� ����� � ������ ������� 
	for (int s = blockDim.x / 2; s > 0; s = s / 2) {		// ��������� ���, ����� ���� �� �������� � ����� � ��� �� �������
		if (threadColumn < s) {
			product[threadRow][threadColumn] += product[threadRow][threadColumn + s];
			sqrdiff[threadRow][threadColumn] += sqrdiff[threadRow][threadColumn + s];
		}
		__syncthreads();
	}
	// ��������� �������� ������� �������, �������� ����� ����
	if (threadColumn == 0){
		for (int s = blockDim.y / 2; s > 0; s = s / 2) {		// ��������� ���, ����� ���� �� �������� � ����� � ��� �� �������
			if (threadRow < s) {
				product[threadRow][threadColumn] += product[threadRow + s][threadColumn];
				sqrdiff[threadRow][threadColumn] += sqrdiff[threadRow + s][threadColumn];
			}
			__syncthreads();
		}
	}

	// ����� ������� ������ ���������� ����� ��������� � ���������� � product[0][0] � sqrdiff[0][0]
	numerator = product[0][0];
	denominator = sqrdiff[0][0];

	// ���������� �������� ���� ����������
	if (denominator == 0){
		orientation.SetAt(row, column, M_PI_2);
	}
	else{
		orientation.SetAt(row, column, M_PI_2 + atan2(2 * numerator, denominator) / 2.0f);
		if (orientation.At(row, column) > M_PI_2){
			orientation.SetAt(row, column, orientation.At(row, column) - M_PI);
		}
	}
}

// ----------- CPU ----------- //

void SetOrientationInBlocks(CUDAArray<float> orientation, CUDAArray<float> source, const int defaultBlockSize, CUDAArray<float> gradientX, CUDAArray<float> gradientY){
	dim3 blockSize = dim3(defaultBlockSize, defaultBlockSize);
	dim3 gridSize =
		dim3(ceilMod(source.Width, defaultBlockSize),
		ceilMod(source.Height, defaultBlockSize));
	cudaSetOrientationInBlocks << <gridSize, blockSize >> >(orientation, gradientX, gradientY);
	hipError_t error = hipDeviceSynchronize();
}

void SetOrientationInPixels(CUDAArray<float> orientation, CUDAArray<float> source, CUDAArray<float> gradientX, CUDAArray<float> gradientY){
	dim3 blockSize = dim3(defaultThreadCount, defaultThreadCount);
	dim3 gridSize =
		dim3(ceilMod(source.Width, defaultThreadCount),
		ceilMod(source.Height, defaultThreadCount));
	cudaSetOrientationInPixels << <gridSize, blockSize >> >(orientation, gradientX, gradientY);
	hipError_t error = hipDeviceSynchronize();
	float* o = orientation.GetData();
}


float* OrientationFieldInBlocks(float* floatArray, int width, int height){
	CUDAArray<float> source(floatArray, width, height);
	const int defaultBlockSize = 16;
	CUDAArray<float> Orientation(source.Width, source.Height);

	// ������� ������
	float filterXLinear[9] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
	float filterYLinear[9] = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };
	// ������� ��� �������
	CUDAArray<float> filterX(filterXLinear, 3, 3);
	CUDAArray<float> filterY(filterYLinear, 3, 3);
	
	// ���������
	CUDAArray<float> Gx(width, height);
	CUDAArray<float> Gy(width, height);
	Convolve(Gx, source, filterX);
	Convolve(Gy, source, filterY);

	// ��������� �����������
	SetOrientationInBlocks(Orientation, source, defaultBlockSize, Gx, Gy);
	
	return Orientation.GetData();
}

float* OrientationFieldInPixels(float* floatArray, int width, int height){

	CUDAArray<float> source(floatArray, width, height);
	CUDAArray<float> Orientation(source.Width, source.Height);

	// ������� ������
	float filterXLinear[9] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
	float filterYLinear[9] = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };
	// ������� ��� �������
	CUDAArray<float> filterX(filterXLinear, 3, 3);
	CUDAArray<float> filterY(filterYLinear, 3, 3);

	// ���������
	CUDAArray<float> Gx(width, height);
	CUDAArray<float> Gy(width, height);
	Convolve(Gx, source, filterX);
	Convolve(Gy, source, filterY);

	SetOrientationInPixels(Orientation, source, Gx, Gy);
	
	return Orientation.GetData();
}

void OrientationFieldInPixels(float* res, float* floatArray, int width, int height){

	CUDAArray<float> source(floatArray, width, height);
	CUDAArray<float> Orientation(source.Width, source.Height);

	// ������� ������
	float filterXLinear[9] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
	float filterYLinear[9] = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };
	// ������� ��� �������
	CUDAArray<float> filterX(filterXLinear, 3, 3);
	CUDAArray<float> filterY(filterYLinear, 3, 3);

	// ���������
	CUDAArray<float> Gx(width, height);
	CUDAArray<float> Gy(width, height);
	Convolve(Gx, source, filterX);
	Convolve(Gy, source, filterY);

	SetOrientationInPixels(Orientation, source, Gx, Gy);

	Orientation.GetData(res);
}



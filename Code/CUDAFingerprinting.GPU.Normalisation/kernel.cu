#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <math.h>
#include "CUDAArray.cuh"
__global__ void cudaCalcMeanRow(CUDAArray<float> source, float* meanArray)
{

	int row = defaultRow();
	int height = source.Height;
	int width = source.Width;
	int tempIndex = threadIdx.x;

	__shared__ float* temp;
	temp = (float*)malloc(blockDim.x * sizeof(float));
	float mean = 0;
	if (source.Height > row)
	{
		for (int j = 0; j < source.Width; j++)
		{
			mean += source.At(row, j) / (height * width);
		}
	}
	temp[tempIndex] = mean;
	__syncthreads();

	//This is reduction.It will work only if number of threads in the block is a power of 2.
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (tempIndex < i)
			temp[tempIndex] += temp[tempIndex + i];
		__syncthreads();
		i /= 2;
	}
	if (tempIndex == 0)
		meanArray = temp;//we need to write it only one time. Why not to choose the first thread for this purpose?
}

float CalculateMean(CUDAArray<float> image)
{
	int height = image.Height;
	float *dev_mean, mean = 0;

	dim3 blockSize = dim3(defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(height, defaultThreadCount));
	float* meanArray = (float*)malloc(gridSize.x * sizeof(float));

	cudaCalcMeanRow << <gridSize, blockSize >> > (image, dev_mean);
	hipMemcpy(meanArray, dev_mean, gridSize.x * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < height; i++)
	{
		mean += meanArray[i];
	}
	return mean;
}

__global__ void cudaCalcVariationRow(CUDAArray<float> image, float mean, float* variationArray)
{

	int row = defaultRow();
	int height = image.Height;
	int width = image.Width;
	int tempIndex = threadIdx.x;

	__shared__ float* temp;
	temp = (float*)malloc(blockDim.x * sizeof(float));
	float variation = 0;
	if (image.Height > row)
	{
		for (int j = 0; j < image.Width; j++)
		{
			variation += pow((image.At(row, j) - mean), 2) / (height * width);
		}
	}
	temp[tempIndex] = variation;
	__syncthreads();
	//This is reduction.It will work only if number of threads in the block is a power of 2.
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (tempIndex < i)
			temp[tempIndex] += temp[tempIndex + i];
		__syncthreads();
		i /= 2;
	}
	if (tempIndex == 0)
		variationArray = temp;//we need to write it only one time. Why not to choose the first thread for this purpose?
}

float CalculateVariation(CUDAArray<float> image, float mean)
{
	int height = image.Height;
	float *dev_variation, variation = 0;

	dim3 blockSize = dim3(defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(height, defaultThreadCount));
	float* variationArray = (float*)malloc(gridSize.x * sizeof(float));

	cudaCalcVariationRow << <gridSize, blockSize >> > (image, mean, dev_variation);
	hipMemcpy(variationArray, dev_variation, gridSize.x * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < height; i++)
	{
		variation += variationArray[i];
	}
	return variation;
}
__global__ void cudaDoNormalizationRow(CUDAArray<float> image, float mean, float variation, int bordMean, int bordVar)
{
	int row = defaultRow();

	if (image.Height > row)
	{
		for (int j = 0; j < image.Width; j++)
		{
			if (image.At(row, j) > mean)
			{
				image.SetAt(row, j, bordMean + sqrt((bordVar * pow(image.At(row, j) - mean, 2)) / variation));
			}
			else
			{
				image.SetAt(row, j, bordMean - sqrt((bordVar * pow(image.At(row, j) - mean, 2)) / variation));
			}
		}
	}
}

CUDAArray<float> DoNormalization(CUDAArray<float> image, int bordMean, int bordVar)
{
	int height = image.Height;

	float mean = CalculateMean(image);
	float variation = CalculateVariation(image, mean);

	dim3 blockSize = dim3(defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(height, defaultThreadCount));
	cudaDoNormalizationRow << <gridSize, blockSize >> > (image, mean, variation, bordMean, bordVar);
	return image;
}
void main()
{
}
/*

CUDAArray<float> DoNormalization(CUDAArray<float> image, int bordMean, int bordVar)
{
float mean = CalculateMean(image);
float variation = CalculateVariation(image, mean);

for (int i = 0; i < image.Width; i++)
{
for (int j = 0; j < image.Height; j++)
{
if (image.At(i, j) > mean)
{
image.SetAt(i, j, bordMean + sqrt((bordVar * pow(image.At(i, j) - mean, 2)) / variation));
}
else
{
image.SetAt(i, j), bordMean - sqrt((bordVar * pow(image.At(i, j) - mean, 2)) / variation));
}
}
}

return image;
float CalculateVariation(CUDAArray<float> image, float mean)
{
int height = image.Height;
int width = image.Width;
float variation = 0;
for (int i = 0; i < width; i++)
{
for (int j = 0; j < height; j++)
{
variation += pow((image.At(i, j) - mean), 2) / (height * width);
}
}
return variation;
}

*/
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/
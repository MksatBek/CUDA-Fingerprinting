#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "constsmacros.h"
#include "BinTemplateCorrelation.cu"
#include "CylinderHelper.cuh"
#include "ConvexHull.cuh"
#include "CUDAArray.cuh"
#include "hip/hip_math_constants.h"
#include "VectorHelper.cuh"
#include "math.h"
#include "math.h"
#include "TemplateCreation.cuh"
#include "device_functions_decls.h"
#include "ConvexHullModified.cuh"
#include <stdio.h>

__device__  Point* getPoint(Minutia *minutiae)
{
	return &Point(
		(float)
		((*minutiae).x + (*constsGPU).baseCell *
		(cos((*minutiae).angle) * (defaultX() - ((*constsGPU).baseCuboid + 1) / 2.0) +
		sin((*minutiae).angle) * (defaultY() - ((*constsGPU).baseCuboid + 1) / 2.0))),
		(float)
		((*minutiae).y + (*constsGPU).baseCell *
		(-sin((*minutiae).angle) * (defaultX() - ((*constsGPU).baseCuboid + 1) / 2.0) +
		cos((*minutiae).angle) * (defaultY() - ((*constsGPU).baseCuboid + 1) / 2.0)))
		);
}

/*__device__ void getNeighborhood(CUDAArray<Minutia> *minutiaArr, Minutia** neighborhood, int* lenghtNeighborhood)
{
	int validMinutiaeLenght = 0;
	Minutia* tmp[200];
	for (size_t i = 0; i < (*minutiaArr).Height*(*minutiaArr).Width; i++)
	{
		if ((pointDistance(Point((float)(*minutiaArr).At(0, i).x, (float)((*minutiaArr).At(0, i).y)), 
			*getPoint(&(*minutiaArr).At(0, defaultMinutia())))) < 3 * (*constsGPU).sigmaLocation &&
			(!equalsMinutae((*minutiaArr).AtPtr(0, i), (*minutiaArr).AtPtr(0, defaultMinutia()))))
		{
			tmp[validMinutiaeLenght] = &((*minutiaArr).At(0, i));
			validMinutiaeLenght++;
		}
	}
	for (size_t i = 0; i < validMinutiaeLenght; i++)
	{
		neighborhood[i] = tmp[i];
		(*lenghtNeighborhood)++;
	}
}*/


__device__  float angleHeight()
{
	return (-HIP_PI + (defaultZ() - 0.5) * (*constsGPU).heightCell);
}

__device__  float gaussian1D(float x)
{
	return expf(-(x * x) / (2 * (*constsGPU).sigmaLocation * (*constsGPU).sigmaLocation)) / ((*constsGPU).sigmaLocation * sqrtf(HIP_PI * 2));
}
/*not work
__device__  float gaussianLocation(Minutia *minutia, Point *point)
{
	return gaussian1D(pointDistance(Point((*minutia).x, (*minutia).y), *point));
}
*/

__device__ float gaussianDirection(Minutia *middleMinutia, Minutia *minutia, float anglePoint)
{
	float common = sqrt(2.0) * (*constsGPU).sigmaDirection;
	double angle = getAngleDiff(anglePoint,
		getAngleDiff((*middleMinutia).angle, (*minutia).angle));
	double first = erf(((angle + (*constsGPU).heightCell / 2)) / common);
	double second = erf(((angle - (*constsGPU).heightCell / 2)) / common);
	return (first - second) / 2;
}

__inline__ __device__ bool equalsMinutae(Minutia* firstMinutia, Minutia* secondMinutia)
{
	return (
		(*firstMinutia).x == (*secondMinutia).x &&
		(*firstMinutia).y == (*secondMinutia).y &&
		abs((*firstMinutia).angle - (*secondMinutia).angle) < 1.401298E-45
		);
}
/*not work
__device__ bool isValidPoint(Minutia* middleMinutia)
{
	return pointDistance(Point((*middleMinutia).x, (*middleMinutia).y), *getPoint(middleMinutia)) < (*constsGPU).radius &&
		isPointInsideHull(*getPoint(middleMinutia), hullGPU, *hullLenghtGPU);
}
*/
/*not work
__device__ float sum(Minutia** neighborhood, Minutia* middleMinutia, int lenghtNeigborhood)
{
	double sum = 0;
	for (size_t i = 0; i < lenghtNeigborhood; i++)
	{
		sum += gaussianLocation(&(*neighborhood[i]), getPoint(middleMinutia)) * gaussianDirection(middleMinutia, neighborhood[i], angleHeight());
	}
	return sum;
}
*/

__device__ char stepFunction(float value)
{
	return (char)(value >= (*constsGPU).sigmoidParametrPsi ? 1 : 0);
}

__global__ void getPoints(CUDAArray<Minutia> minutiae, CUDAArray<Point> points)
{
	if (threadIdx.x < minutiae.Width)
	{
		points.SetAt(0, threadIdx.x, Point(minutiae.At(0, threadIdx.x).x, minutiae.At(0, threadIdx.x).y));
	}
}

__global__ void getValidMinutiae(CUDAArray<Minutia> minutiae, CUDAArray<bool> isValidMinutiae)
{
	if (threadIdx.x >= minutiae.Width)
	{
		return;
	}
	int validMinutiaeLenght = 0;
	for (int i = 0; i < minutiae.Width; i++)
	{
		if (threadIdx.x == i)
		{
			continue;
		}
		validMinutiaeLenght = sqrt((float)
			((minutiae.At(0, threadIdx.x).x - minutiae.At(0, i).x)*(minutiae.At(0, threadIdx.x).x - minutiae.At(0, i).x) +
			minutiae.At(0, threadIdx.x).y - minutiae.At(0, i).y)*(minutiae.At(0, threadIdx.x).y - minutiae.At(0, i).y))
			< (*constsGPU).radius + 3 * (*constsGPU).sigmaLocation ? validMinutiaeLenght + 1 : validMinutiaeLenght;
	}
	isValidMinutiae.SetAt(0, threadIdx.x, validMinutiaeLenght >= (*constsGPU).minNumberMinutiae ? true : false);
}

__global__ void createSum(CUDAArray<unsigned int> valuesAndMasks, CUDAArray<unsigned int> sum)
{
	unsigned int x = __popc(valuesAndMasks.At(defaultMinutia(), threadIdx.x * 2 + blockIdx.x));
	atomicAdd(sum.AtPtr(0, threadIdx.x * 2 + blockIdx.x), x);
}


__global__ void createCylinders(CUDAArray<Minutia> minutiae, CUDAArray<unsigned int> sum, 
	CUDAArray<unsigned int> valuesAndMasks, CUDAArray<Cylinder> cylinders)
{
	cylinders.SetAt(0, blockIdx.x, Cylinder(valuesAndMasks.AtPtr(blockIdx.x, 0), valuesAndMasks.Width,
		minutiae.At(0, blockIdx.x).angle, sqrt((float)(sum.At(0, blockIdx.x))), 0));
}
/*not work
__global__ void createValuesAndMasks(CUDAArray<Minutia> minutiae, CUDAArray<unsigned int> valuesAndMasks, Minutia ** neighborhood)
{
	int lenghtNeighborhood = 0;
	if (defaultX() > 16 || defaultY() > 16 || defaultZ() > 6 || defaultMinutia() > minutiae.Width)
	{
		return;
	}
	if (isValidPoint(&minutiae.At(0, defaultMinutia())))
	{
		char tempValue =
			(defaultY() % 2)*(stepFunction(sum(getNeighborhood(&minutiae, &lenghtNeighborhood), &(minutiae.At(0, defaultMinutia())), lenghtNeighborhood)));
		atomicOr(valuesAndMasks.AtPtr(defaultMinutia(), curIndex()), (tempValue - '0' + blockIdx.y) << linearizationIndex() % 32);
	}
	else
	{
		atomicOr(valuesAndMasks.AtPtr(defaultMinutia(), curIndex()), 0 << linearizationIndex() % 32);
	}
}
*/
int main()
{
   
}

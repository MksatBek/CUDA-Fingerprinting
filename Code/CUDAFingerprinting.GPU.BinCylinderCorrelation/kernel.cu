#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "constsmacros.h"
#include "CUDAArray.cuh"
#include <cstring>
#include <time.h>

extern "C"
{
	__declspec(dllexport) float getBinCylinderCorrelation(
		unsigned int cylinderCapacity,
		unsigned int *cudaCylinder1, unsigned int *cudaCylinder2,
		unsigned int *cudaValidities1, unsigned int *cudaValidities2);
}

void printArray1D(unsigned int* arr, unsigned int length)
{
	for (unsigned int i = 0; i < length; i++) {
		printf("%u ", arr[i]);
	}
	printf("\n");
}

void printCUDAArray1D(CUDAArray<unsigned int> arr)
{
	printf("Print CUDAArray\n");
	printArray1D(arr.GetData(), arr.Width * arr.Height);
	printf("[end] Print CUDAArray\n");
}

__global__ void cudaArrayBitwiseAnd(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd, CUDAArray<unsigned int> result)
{
	int row = defaultRow();
	int column = defaultColumn();

	if (fst.Width > column && fst.Height > row) {
		unsigned int newValue = fst.At(row, column) & snd.At(row, column);
		result.SetAt(row, column, newValue);
	}
}

CUDAArray<unsigned int> BitwiseAndArray(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd)
{
	dim3 gridSize = dim3(ceilMod(fst.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	CUDAArray<unsigned int> *result = new CUDAArray<unsigned int>(fst.Width, 1);

	cudaArrayBitwiseAnd << <gridSize, blockSize >> >(fst, snd, *result);

	return *result;
}


__global__ void cudaArrayBitwiseXor(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd, CUDAArray<unsigned int> result)
{
	int row = defaultRow();
	int column = defaultColumn();

	if (fst.Width > column && fst.Height > row)	{
		unsigned int newValue = fst.At(row, column) ^ snd.At(row, column);
		result.SetAt(row, column, newValue);
	}
}

CUDAArray<unsigned int> BitwiseXorArray(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd)
{
	dim3 gridSize = dim3(ceilMod(fst.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	CUDAArray<unsigned int> *result = new CUDAArray<unsigned int>(fst.Width, 1);

	cudaArrayBitwiseXor << <gridSize, blockSize >> >(fst, snd, *result);

	//printCUDAArray1D(*result);

	return *result;
}

__global__ void cudaArrayWordNorm(CUDAArray<unsigned int> arr, unsigned int* sum)
{
	int row = defaultRow();
	int column = defaultColumn();

	if (arr.Width > column && arr.Height > row)	{
		unsigned int x = arr.At(row, column);

		x = x - ((x >> 1) & 0x55555555);
		x = (x & 0x33333333) + ((x >> 2) & 0x33333333);
		x = (x + (x >> 4)) & 0x0F0F0F0F;
		x = x + (x >> 8);
		x = x + (x >> 16);
		x = x & 0x0000003F;

		atomicAdd(sum, x);
	}
}

unsigned int getOneBitsCount(CUDAArray<unsigned int> arr)
{
	dim3 gridSize = dim3(ceilMod(arr.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	unsigned int* sum = (unsigned int*)malloc(sizeof(unsigned int));
	*sum = 0;

	unsigned int* d_sum;
	hipMalloc((unsigned int **)&d_sum, sizeof(unsigned int));
	hipMemcpy(d_sum, sum, sizeof(unsigned int), hipMemcpyHostToDevice);

	cudaArrayWordNorm << <gridSize, blockSize >> >(arr, d_sum);

	hipMemcpy(sum, d_sum, sizeof(unsigned int), hipMemcpyDeviceToHost);

	return *sum;
}

float getBinCylinderCorrelation(
	unsigned int cylinderCapacity,
	unsigned int *cudaCylinder1, unsigned int *cudaCylinder2,
	unsigned int *cudaValidities1, unsigned int *cudaValidities2)
{
	//printArray1D(cudaValidities1, cylinderCapacity);

	CUDAArray<unsigned int> *linearizedCylinder1 = new CUDAArray<unsigned int>(cudaCylinder1, cylinderCapacity, 1);
	CUDAArray<unsigned int> *linearizedCylinder2 = new CUDAArray<unsigned int>(cudaCylinder2, cylinderCapacity, 1);

	CUDAArray<unsigned int> *cylinder1Validities = new CUDAArray<unsigned int>(cudaValidities1, cylinderCapacity, 1);
	CUDAArray<unsigned int> *cylinder2Validities = new CUDAArray<unsigned int>(cudaValidities2, cylinderCapacity, 1);

	CUDAArray<unsigned int> commonValidities = BitwiseAndArray(*cylinder1Validities, *cylinder2Validities);
	CUDAArray<unsigned int> c1GivenCommon = BitwiseAndArray(*linearizedCylinder1, commonValidities);
	CUDAArray<unsigned int> c2GivenCommon = BitwiseAndArray(*linearizedCylinder2, commonValidities);

	//printf("///\n");
	//printCUDAArray1D(cylinder1Validities);
	//printCUDAArray1D(commonValidities);
	//printCUDAArray1D(c1GivenCommon);
	//printCUDAArray1D(c2GivenCommon);
	//printCUDAArray1D(linearizedCylinder2);
	//printf("///\n");

	unsigned int c1GivenCommonOnesCount = getOneBitsCount(c1GivenCommon);
	float c1GivenCommonNorm = sqrt((float)c1GivenCommonOnesCount);
	unsigned int c2GivenCommonOnesCount = getOneBitsCount(c2GivenCommon);
	float c2GivenCommonNorm = sqrt((float)c2GivenCommonOnesCount);

	//printf("%u, %u\n", c1GivenCommonOnesCount, c2GivenCommonOnesCount);
	//printf("%f, %f\n", c1GivenCommonNorm, c2GivenCommonNorm);

	bool matchable = true;

	if (c1GivenCommonNorm + c2GivenCommonNorm == 0) {
		matchable = false;
	}

	float correlation = 0.0f;

	if (matchable) {
		CUDAArray<unsigned int> givenXOR = BitwiseXorArray(c1GivenCommon, c2GivenCommon);
		//printCUDAArray1D(givenXOR);
		unsigned int givenXORBytesCount = getOneBitsCount(givenXOR);
		float givenXORNorm = sqrt((float)givenXORBytesCount);
		correlation = 1 - givenXORNorm / (c1GivenCommonNorm + c2GivenCommonNorm);
	}

	delete(linearizedCylinder1);
	delete(linearizedCylinder2);
	delete(cylinder1Validities);
	delete(cylinder2Validities);

	return correlation;
}

unsigned int binToInt(char* s)
{
	return (unsigned int)strtoul(s, NULL, 2);
}

int main()
{
	unsigned int cylinderCapacity = 1;

	unsigned int *cudaCylinder1 = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));
	unsigned int *cudaCylinder2 = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));

	unsigned int *cudaValidities1 = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));
	unsigned int *cudaValidities2 = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));

	// Test 1
	//memset(cudaCylinder1, 255, cylinderCapacity * sizeof(unsigned int));
	//memset(cudaCylinder2, 255, cylinderCapacity * sizeof(unsigned int));
	//memset(cudaValidities1, 255, cylinderCapacity * sizeof(unsigned int));
	//memset(cudaValidities2, 255, cylinderCapacity * sizeof(unsigned int));
	//getBinCylinderCorrelation(cylinderCapacity, cudaCylinder1, cudaCylinder2, cudaValidities1, cudaValidities2);


	// Test 2
	//srand((unsigned int)time(NULL));
	//for (unsigned int i = 0; i < cylinderCapacity; i++) {
	//	cudaCylinder1[i] = rand();
	//	cudaCylinder2[i] = rand();
	//	cudaValidities1[i] = rand();
	//	cudaValidities2[i] = rand();
	//}
	//getBinCylinderCorrelation(cylinderCapacity, cudaCylinder1, cudaCylinder2, cudaValidities1, cudaValidities2);

	// Test 3 (only for cylinderCapacity == 1)

	cudaCylinder1[0] = binToInt("11111111111111111100000000000000");
	cudaValidities1[0] = binToInt("11111111111111111100000000000000");

	cudaCylinder2[0] = binToInt("11010001010100001100000000000000");
	cudaValidities2[0] = binToInt("11011101111100011100000000000000");

	float correlation =
		getBinCylinderCorrelation(cylinderCapacity, cudaCylinder1, cudaCylinder2, cudaValidities1, cudaValidities2);

	printf("Correlation: %f\n", correlation);

	// [end] Test 3

	free(cudaCylinder1);
	free(cudaCylinder2);
	free(cudaValidities1);
	free(cudaValidities2);

	return 0;
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include "constsmacros.h"
#include "CUDAArray.cuh"
#include <cstring>
#include <time.h>
#include "CylinderHelper.cuh"
#include <hip/hip_math_constants.h>
#include <math.h>
#include "BinCorrelationHelper.cuh"
#include "BinTemplateCorrelation.cuh"

#define MIN(a, b) ((a) < (b) ? (a) : (b))

#define MAX_CYLINDERS_PER_TEMPLATE 256
//#define CYLINDER_CELLS_COUNT 255 // Hopefully this define is not necessary

#define QUANTIZED_ANGLES_COUNT 256
#define QUANTIZED_SIMILARITIES_COUNT 64 // Basically buckets count

#define END_OF_LIST -1

#define ANGLE_THRESHOLD 0.52359877f

#define THREADS_PER_BLOCK_MATRIX_GEN 192
#define THREADS_PER_BLOCK_LSS 64

#define NUM_PAIRS_MIN 11
#define NUM_PAIRS_MAX 13
#define NUM_PAIRS_MU 30
#define NUM_PAIRS_TAU 0.4

__constant__ CUDAArray<CylinderGPU> queryGPU;
__device__ CUDAArray<CylinderGPU> cylindersDbGPU;
__device__ CUDAArray<unsigned int> bucketMatrix;
__constant__ unsigned int queryLength;

float getAngleDiff(float angle1, float angle2)
{
	float diff = angle1 - angle2;
	return
		diff < -HIP_PI_F ? diff + 2 * HIP_PI_F :
		diff >= HIP_PI_F ? diff - 2 * HIP_PI_F :
		diff;
}


__global__ void computeLUTSqrt(CUDAArray<float> LUTSqrt)
{
	LUTSqrt.SetAt(threadIdx.x, 1, sqrt((float)threadIdx.x));
}

__global__ void computeLUTNumPairs(CUDAArray<unsigned int> LUTNumPairs)
{
	for (int i = 0; i < LUTNumPairs.Width; i++)
	{
		unsigned int curNumPairs = NUM_PAIRS_MIN + round((NUM_PAIRS_MAX - NUM_PAIRS_MIN)
			/ (1 + expf(-NUM_PAIRS_TAU * MIN(queryLength, i) - NUM_PAIRS_MU)));
		LUTNumPairs.SetAt(i, 1, curNumPairs);
	}
}

__global__ void computeLUTAngles(CUDAArray<unsigned int> LUTAngles)
{
	float curAngle = 2 * HIP_PI_F * threadIdx.x / QUANTIZED_ANGLES_COUNT;
	int i;
	for (i = 0; i < queryGPU.Width; i++)
	{
		if (getAngleDiff(queryGPU.At(i, 1).angle, curAngle) < ANGLE_THRESHOLD
			&& queryGPU.At(i, 1).norm + queryGPU.At(i, 1).norm != 0)
		{
			LUTAngles.SetAt(curAngle, i, i);
		}
	}
	LUTAngles.SetAt(curAngle, i, END_OF_LIST);
}

__global__ void generateBucketMatrix(CUDAArray<float> LUTSqrt, CUDAArray<unsigned int> LUTAngles)
{
	int row = defaultRow();
	int column = defaultRow();

	CylinderGPU curCylinderDb = cylindersDbGPU.At(row, column);
	float curCylinderDbAngle = cylindersDbGPU.At(row, column).angle;

	for (int i = 0; LUTAngles.At(curCylinderDbAngle, i) != -1; i++)
	{
		CylinderGPU curCylinderQuery = queryGPU.At(i, 1);
		CUDAArray<unsigned int> valuesDb = curCylinderDb.values;
		CUDAArray<unsigned int> valuesQuery = curCylinderQuery.values;

		unsigned int bucketIndex = (unsigned int)floor(
			LUTSqrt.At(getOneBitsCount(BitwiseXorArray(valuesDb, valuesQuery)), 1)
			/ (curCylinderDb.norm + curCylinderQuery.norm) * QUANTIZED_SIMILARITIES_COUNT);

		bucketMatrix.SetAt(curCylinderDb.templateIndex, bucketIndex, bucketMatrix.At(curCylinderDb.templateIndex, bucketIndex) + 1);
	}
}

__global__ void computeLSS(
	CUDAArray<unsigned int> LUTTemplateDbLengths, CUDAArray<unsigned int>LUTNumPairs, CUDAArray<unsigned int> similarityRates)
{
	// Index of the first DB template of the current block
	unsigned int firstBlockTemplateIndex = THREADS_PER_BLOCK_LSS * blockIdx.x;
	__shared__ CUDAArray<unsigned int> bucketSubmatrixPerBlock(QUANTIZED_SIMILARITIES_COUNT, THREADS_PER_BLOCK_LSS);

	// Copy appropriate part of bucketMatrix to bucketSubmatrixPerBlock
	for (int j = 0; j < THREADS_PER_BLOCK_LSS; j++)
	{
		bucketSubmatrixPerBlock.SetAt(j, threadIdx.x, bucketMatrix.At(firstBlockTemplateIndex + j, threadIdx.x));
	}

	__syncthreads();

	unsigned int curTemplateIndex = firstBlockTemplateIndex + threadIdx.x; // == defaultColumn() (hopefully)

	unsigned int numPairs = LUTNumPairs.At(MIN(LUTTemplateDbLengths.At(curTemplateIndex, 1), queryLength), 1);
	int sum = 0, t = numPairs, i = 0;
	while (i < QUANTIZED_SIMILARITIES_COUNT && t > 0)
	{
		unsigned int curBucketValue = bucketSubmatrixPerBlock.At(threadIdx.x, i);
		sum += MIN(curBucketValue, t);
		t -= MIN(curBucketValue, t);
		i++;
	}
	sum += t * QUANTIZED_SIMILARITIES_COUNT;

	similarityRates.SetAt(curTemplateIndex, 1, 1 - sum / (numPairs * QUANTIZED_SIMILARITIES_COUNT));
}

void convertToCylindersGPU(Cylinder *cylinders, unsigned int cylindersCount, CUDAArray<CylinderGPU> cylindersGPU)
{
	for (int i = 0; i < cylindersCount; i++)
	{
		Cylinder *curCylinder = &(cylinders[i]);
		CylinderGPU *curCylinderGPU = new CylinderGPU(
			curCylinder->values, curCylinder->valuesCount, curCylinder->angle, curCylinder->norm);
		cylindersGPU.SetAt(i, 1, *curCylinderGPU);
	}
}

void getBinTemplateSimilarity(
	Cylinder *query, unsigned int queryLength,
	Cylinder *cylindersDb, unsigned int cylinderDbCount,
	unsigned int *templateDbLengths, unsigned int templateDbCount,
	float *similarityRates)
{
	queryGPU = CUDAArray<CylinderGPU>(queryLength, 1);
	convertToCylindersGPU(query, queryLength, queryGPU);

	cylindersDbGPU = CUDAArray<CylinderGPU>(cylinderDbCount, 1);
	convertToCylindersGPU(cylindersDb, cylinderDbCount, cylindersDbGPU);

	CUDAArray<unsigned int> LUTTemplateDbLengths(templateDbLengths, templateDbCount, 1);

	// It's supposed to work only when all the cylinders have the same length, index = 0 WLOG
	unsigned int cylinderCellsCount = cylindersDb[0].valuesCount;

	// 0 through cylinderCellsCount (population count values)
	CUDAArray<float> LUTSqrt(cylinderCellsCount + 1, 1);
	computeLUTSqrt << <1, CYLINDER_ELEMENTS_COUNT + 1 >> >(LUTSqrt);

	CUDAArray<unsigned int> LUTNumPairs(cylinderCellsCount, 1);
	computeLUTNumPairs << <1, cylinderCellsCount >> >(LUTNumPairs);

	CUDAArray<unsigned int> LUTAngles(QUANTIZED_ANGLES_COUNT, queryLength + 1);
	computeLUTAngles << <1, QUANTIZED_ANGLES_COUNT >> >(LUTAngles);

	bucketMatrix = CUDAArray<unsigned int>(QUANTIZED_SIMILARITIES_COUNT, templateDbCount);
	generateBucketMatrix << <ceilMod(cylinderDbCount, THREADS_PER_BLOCK_MATRIX_GEN), THREADS_PER_BLOCK_MATRIX_GEN >> >(LUTSqrt, LUTAngles);

	CUDAArray<float> similaritiesVector(templateDbCount, 1);
	computeLSS << <ceilMod(templateDbCount, THREADS_PER_BLOCK_LSS), THREADS_PER_BLOCK_LSS >> >
		(LUTTemplateDbLengths, LUTNumPairs, similaritiesVector);

	similarityRates = similaritiesVector.GetData(); // Assigning the result

	LUTSqrt.Dispose();
	LUTAngles.Dispose();
	LUTNumPairs.Dispose();
	bucketMatrix.Dispose();
	similaritiesVector.Dispose();
}

unsigned int binToInt(char* s)
{
	return (unsigned int)strtoul(s, NULL, 2);
}

int main()
{
	unsigned int cylinderCapacity = 1;

	unsigned int *cudaCylinder1 = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));
	unsigned int *cudaCylinder2 = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));

	unsigned int *cudaValidities1 = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));
	unsigned int *cudaValidities2 = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));

	// Test 1
	//memset(cudaCylinder1, 255, cylinderCapacity * sizeof(unsigned int));
	//memset(cudaCylinder2, 255, cylinderCapacity * sizeof(unsigned int));
	//memset(cudaValidities1, 255, cylinderCapacity * sizeof(unsigned int));
	//memset(cudaValidities2, 255, cylinderCapacity * sizeof(unsigned int));
	//getBinCylinderCorrelation(cylinderCapacity, cudaCylinder1, cudaCylinder2, cudaValidities1, cudaValidities2);


	// Test 2
	//srand((unsigned int)time(NULL));
	//for (unsigned int i = 0; i < cylinderCapacity; i++) {
	//	cudaCylinder1[i] = rand();
	//	cudaCylinder2[i] = rand();
	//	cudaValidities1[i] = rand();
	//	cudaValidities2[i] = rand();
	//}
	//getBinCylinderCorrelation(cylinderCapacity, cudaCylinder1, cudaCylinder2, cudaValidities1, cudaValidities2);

	// Test 3 (only for cylinderCapacity == 1)

	cudaCylinder1[0] = binToInt("11111111111111111100000000000000");
	cudaValidities1[0] = binToInt("11111111111111111100000000000000");

	cudaCylinder2[0] = binToInt("11010001010100001100000000000000");
	cudaValidities2[0] = binToInt("11011101111100011100000000000000");

	float correlation =
		getBinCylinderCorrelation(cylinderCapacity, cudaCylinder1, cudaCylinder2, cudaValidities1, cudaValidities2);

	printf("Correlation: %f\n", correlation);

	// [end] Test 3

	free(cudaCylinder1);
	free(cudaCylinder2);
	free(cudaValidities1);
	free(cudaValidities2);

	return 0;
}
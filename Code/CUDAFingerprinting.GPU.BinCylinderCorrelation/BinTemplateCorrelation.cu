#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include "constsmacros.h"
#include "CUDAArray.cuh"
#include <cstring>
#include <time.h>
#include "CylinderHelper.cuh"
#include <hip/hip_math_constants.h>
#include <math.h>
#include "BinCorrelationHelper.cuh"
#include "BinTemplateCorrelation.cuh"

#define cudaCheckError() {\
	hipError_t e = hipGetLastError(); \
if (e != hipSuccess) {\
	printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
	exit(0); \
}\
}

// Getting element of CUDAArray of unsigned integers without .At() (used with bucketMatrix)
#define UINT_ARRAY_AT(cylinder, row, column) (cylinder.cudaPtr[row * (cylinder.Stride / sizeof(unsigned int)) + column])
// Same with CUDAArray of cylinders (used with xorArray)
#define CYLINDER_ARRAY_AT(arr, row, column) (arr.cudaPtr[row * (arr.Stride / sizeof(CUDAArray<unsigned int>)) + column])

#define MIN(a, b) ((a) < (b) ? (a) : (b))

#define MAX_CYLINDERS_PER_TEMPLATE 256
#define CYLINDER_CELLS_COUNT 1 // Hopefully this define is not necessary (constant memory again)

#define QUANTIZED_ANGLES_COUNT 256
#define QUANTIZED_SIMILARITIES_COUNT 64 // Basically buckets count

#define END_OF_LIST -1

#define ANGLE_THRESHOLD 0.52359877f // == PI / 6

#define THREADS_PER_BLOCK_MATRIX_GEN 192
#define THREADS_PER_BLOCK_LSS 64

#define NUM_PAIRS_MIN 11
#define NUM_PAIRS_MAX 13
#define NUM_PAIRS_MU 30
#define NUM_PAIRS_TAU 0.4

__constant__ CUDAArray<CylinderGPU> queryGPU;
__device__ CUDAArray<CylinderGPU> cylinderDbGPU;
__device__ CUDAArray<unsigned int> bucketMatrix;
__constant__ unsigned int queryLengthGlobal;

__device__ float getAngleDiff(float angle1, float angle2)
{
	float diff = angle1 - angle2;
	return
		diff < -HIP_PI_F ? diff + 2 * HIP_PI_F :
		diff >= HIP_PI_F ? diff - 2 * HIP_PI_F :
		diff;
}


__global__ void computeLUTSqrt(CUDAArray<float> LUTSqrt)
{
	LUTSqrt.SetAt(0, threadIdx.x, sqrt((float)threadIdx.x));
}

__global__ void computeLUTNumPairs(CUDAArray<unsigned int> LUTNumPairs)
{
	int threadIndex = defaultColumn();
	unsigned int curNumPairs = NUM_PAIRS_MIN + round((NUM_PAIRS_MAX - NUM_PAIRS_MIN)
		/ (1 + expf(-1 * NUM_PAIRS_TAU * ((int)MIN(queryLengthGlobal, threadIndex) - NUM_PAIRS_MU)))); // Amazingly important cast
	LUTNumPairs.SetAt(0, threadIndex, curNumPairs);
}

__global__ void computeLUTAngles(CUDAArray<int> LUTAngles)
{
	float curAngle = 2 * HIP_PI_F * threadIdx.x / QUANTIZED_ANGLES_COUNT;
	unsigned int i;
	unsigned int LUTIndex = 0;
	for (i = 0; i < queryGPU.Width; i++)
	{
		if (getAngleDiff(queryGPU.At(i, 0).angle, curAngle) < ANGLE_THRESHOLD)
		{
			LUTAngles.SetAt(curAngle, LUTIndex, i);
			int num = LUTAngles.At(curAngle, LUTIndex);
			LUTIndex++;
		}
	}
	LUTAngles.SetAt(threadIdx.x, LUTIndex, END_OF_LIST);
}

__global__ void computeXorArray(CUDAArray<CUDAArray<unsigned int>> xorArray)
{
	unsigned int threadIndex = defaultColumn();

	if (xorArray.Width * xorArray.Height * CYLINDER_CELLS_COUNT > threadIndex)
	{
		unsigned int xorArrayRow = threadIndex / (queryLengthGlobal * CYLINDER_CELLS_COUNT);
		unsigned int xorArrayColumn = (threadIndex / queryLengthGlobal) % CYLINDER_CELLS_COUNT;

		cudaArrayBitwiseXorDevice(
			cylinderDbGPU.At(0, xorArrayRow).values,
			queryGPU.At(0, xorArrayColumn).values,
			&CYLINDER_ARRAY_AT(xorArray, xorArrayRow, xorArrayColumn));
	}
}

__global__ void cumputeLUTPopCountXor(unsigned int *LUTArr, CUDAArray<CUDAArray<unsigned int>> xorArray)
{
	unsigned int threadIndex = defaultColumn();

	if (xorArray.Width * xorArray.Height * CYLINDER_CELLS_COUNT > threadIndex)
	{
		unsigned int xorArrayRow = threadIndex / (queryLengthGlobal * CYLINDER_CELLS_COUNT);
		unsigned int xorArrayColumn = (threadIndex / queryLengthGlobal) % CYLINDER_CELLS_COUNT;


		cudaArrayWordNormDevice(
			&CYLINDER_ARRAY_AT(xorArray, xorArrayRow, xorArrayColumn),
			&LUTArr[xorArrayRow * queryLengthGlobal + xorArrayColumn]);
	}
}

__global__ void generateBucketMatrix(CUDAArray<float> LUTSqrt, CUDAArray<int> LUTAngles, CUDAArray<unsigned int> LUTPopCountXor)
{
	unsigned int curCylinderIndex = defaultColumn();

	if (cylinderDbGPU.Width > curCylinderIndex)
	{
		CylinderGPU curCylinderDb = cylinderDbGPU.At(0, curCylinderIndex);
		float curCylinderDbAngle = curCylinderDb.angle;

		unsigned int angleIndex = (unsigned int)(curCylinderDbAngle * QUANTIZED_ANGLES_COUNT / (2 * HIP_PI_F));
		for (unsigned int i = 0; LUTAngles.At(angleIndex, i) != -1; i++)
		{
			unsigned int curQueryIndex = LUTAngles.At(angleIndex, i);
			CylinderGPU curCylinderQuery = queryGPU.At(0, curQueryIndex);

			unsigned int lutPopCountXor = LUTPopCountXor.At(curCylinderIndex, curQueryIndex);
			float lutSqrt = LUTSqrt.At(0, lutPopCountXor);
			float x = lutSqrt / (curCylinderDb.norm + curCylinderQuery.norm) * QUANTIZED_SIMILARITIES_COUNT; // local similarity inverse (without 1 - ...)
			unsigned int bucketIndex = (unsigned int)floor(x);

			if (bucketIndex == QUANTIZED_SIMILARITIES_COUNT)
			{
				bucketIndex--;
			}

			atomicAdd(&UINT_ARRAY_AT(bucketMatrix, curCylinderDb.templateIndex, bucketIndex), 1);
		}
	}
}

__global__ void computeLSS(
	CUDAArray<unsigned int> LUTTemplateDbLengths, CUDAArray<unsigned int>LUTNumPairs, CUDAArray<float> similarityRates)
{
	// Dynamic allocation will probably be better
	__shared__ unsigned int bucketSubmatrixPerBlock[QUANTIZED_SIMILARITIES_COUNT][THREADS_PER_BLOCK_LSS];

	// Index of the first DB template of the current block
	unsigned int firstBlockTemplateIndex = THREADS_PER_BLOCK_LSS * blockIdx.x;

	// Copy appropriate part of bucketMatrix to bucketSubmatrixPerBlock
	int submatrixHeight =
		QUANTIZED_SIMILARITIES_COUNT % THREADS_PER_BLOCK_LSS == 0 ? THREADS_PER_BLOCK_LSS :
		QUANTIZED_SIMILARITIES_COUNT % THREADS_PER_BLOCK_LSS;
	for (int j = 0; j < submatrixHeight; j++)
	{
		bucketSubmatrixPerBlock[j][threadIdx.x] = bucketMatrix.At(firstBlockTemplateIndex + j, threadIdx.x);
	}

	__syncthreads();

	unsigned int threadIndex = defaultColumn();
	if (bucketMatrix.Height > threadIndex)
	{
		unsigned int numPairs = LUTNumPairs.At(0, MIN(LUTTemplateDbLengths.At(0, threadIndex), queryLengthGlobal));
		int sum = 0, t = numPairs, i = 0;
		while (i < QUANTIZED_SIMILARITIES_COUNT && t > 0)
		{
			unsigned int curBucketValue = bucketSubmatrixPerBlock[threadIdx.x][i];
			sum += MIN(curBucketValue, t) * i;
			t -= MIN(curBucketValue, t);
			i++;
		}
		sum += t * QUANTIZED_SIMILARITIES_COUNT;

		similarityRates.SetAt(0, threadIndex, 1.0 - (float)sum / (numPairs * QUANTIZED_SIMILARITIES_COUNT));
	}
}

void convertToCylindersGPU(Cylinder *cylinders, unsigned int cylindersCount, CUDAArray<CylinderGPU> *cylindersGPU)
{
	CylinderGPU *cylindersGPUarr = (CylinderGPU *)malloc(cylindersCount * sizeof(CylinderGPU));
	for (unsigned int i = 0; i < cylindersCount; i++)
	{
		Cylinder *curCylinder = &(cylinders[i]);
		cylindersGPUarr[i] = CylinderGPU(
			curCylinder->values, curCylinder->valuesCount, curCylinder->angle, curCylinder->norm, curCylinder->templateIndex);
	}

	*cylindersGPU = CUDAArray<CylinderGPU>(cylindersGPUarr, cylindersCount, 1);
}

void printAngles(int* arr, int width, int height)
{
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			printf("%d ", arr[i * width + j]);
		}
		printf("\n");
	}
}

void printCUDAAngles(CUDAArray<int> arr)
{
	printf("Print CUDAArray 2D\n");
	printAngles(arr.GetData(), arr.Width, arr.Height);
	printf("[end] Print CUDAArray 2D\n");
}

float * getBinTemplateSimilarities(
	Cylinder *query, unsigned int queryLength,
	Cylinder *cylinderDb, unsigned int cylinderDbCount,
	unsigned int *templateDbLengths, unsigned int templateDbCount)
{
	hipSetDevice(0);

	hipMemcpyToSymbol(HIP_SYMBOL(queryLengthGlobal), &queryLength, sizeof(unsigned int));
	cudaCheckError();

	CUDAArray<CylinderGPU> preQueryGPU = CUDAArray<CylinderGPU>(queryLength, 1);
	convertToCylindersGPU(query, queryLength, &preQueryGPU);
	hipMemcpyToSymbol(HIP_SYMBOL(queryGPU), &preQueryGPU, sizeof(CUDAArray<CylinderGPU>));
	cudaCheckError();

	CUDAArray<CylinderGPU> preCylinderDbGPU = CUDAArray<CylinderGPU>(cylinderDbCount, 1);
	convertToCylindersGPU(cylinderDb, cylinderDbCount, &preCylinderDbGPU);
	cudaCheckError();
	hipMemcpyToSymbol(HIP_SYMBOL(cylinderDbGPU), &preCylinderDbGPU, sizeof(CUDAArray<CylinderGPU>));
	cudaCheckError();

	CUDAArray<unsigned int> LUTTemplateDbLengths(templateDbLengths, templateDbCount, 1);

	// It's supposed to work only when all the cylinders have the same length, index = 0 WLOG
	unsigned int cylinderCellsCount = cylinderDb[0].valuesCount;

	// 0 through cylinderCellsCount (population count values)
	CUDAArray<float> LUTSqrt = CUDAArray<float>(cylinderCellsCount * sizeof(unsigned int)* 8 + 1, 1); // 0 through number of bits in cylinder
	computeLUTSqrt << <1, cylinderCellsCount * sizeof(unsigned int)* 8 + 1 >> >(LUTSqrt);

	CUDAArray<unsigned int> LUTNumPairs = CUDAArray<unsigned int>(MAX_CYLINDERS_PER_TEMPLATE, 1);
	computeLUTNumPairs << <1, MAX_CYLINDERS_PER_TEMPLATE >> >(LUTNumPairs);

	int *preLUTAngles = (int *)malloc((queryLength + 1) * QUANTIZED_ANGLES_COUNT * sizeof(int));
	memset(preLUTAngles, 0, (queryLength + 1) * QUANTIZED_ANGLES_COUNT * sizeof(int));
	CUDAArray<int> LUTAngles = CUDAArray<int>(preLUTAngles, queryLength + 1, QUANTIZED_ANGLES_COUNT);
	computeLUTAngles << <1, QUANTIZED_ANGLES_COUNT >> >(LUTAngles);

	CUDAArray<unsigned int> *xorArrayCylindersGPU =
		(CUDAArray<unsigned int> *)malloc(cylinderDbCount * queryLength * sizeof(CUDAArray<unsigned int>));
	for (unsigned int i = 0; i < cylinderDbCount * queryLength; i++) {
		xorArrayCylindersGPU[i] = CUDAArray<unsigned int>(1, cylinderCellsCount);
	}
	CUDAArray<CUDAArray<unsigned int>> xorArray = CUDAArray<CUDAArray<unsigned int>>(xorArrayCylindersGPU, queryLength, cylinderDbCount);
	computeXorArray << <ceilMod(cylinderDbCount * queryLength * cylinderCellsCount, defaultThreadCount), defaultThreadCount >> >(xorArray);
	cudaCheckError();
	unsigned int *d_LUTArr;
	hipMalloc((void **)&d_LUTArr, cylinderDbCount * queryLength * sizeof(unsigned int));
	cudaCheckError();
	hipMemset(d_LUTArr, 0, cylinderDbCount * queryLength * sizeof(unsigned int));
	cudaCheckError();
	cumputeLUTPopCountXor << <cylinderDbCount, queryLength * cylinderCellsCount >> >(d_LUTArr, xorArray); // Potentially dangerous (may exceed threads-per-block limitation)
	cudaCheckError();

	unsigned int *h_LUTArr = new unsigned int[cylinderDbCount * queryLength];
	hipMemcpy(h_LUTArr, d_LUTArr, cylinderDbCount * queryLength * sizeof(unsigned int), hipMemcpyDeviceToHost);
	cudaCheckError();

	CUDAArray<unsigned int> LUTPopCountXor = CUDAArray<unsigned int>(h_LUTArr, queryLength, cylinderDbCount);


	unsigned int *zeroMatrix = (unsigned int *)malloc(QUANTIZED_SIMILARITIES_COUNT * templateDbCount * sizeof(unsigned int));
	memset(zeroMatrix, 0, QUANTIZED_SIMILARITIES_COUNT * templateDbCount * sizeof(unsigned int));

	CUDAArray<unsigned int> preBucketMatrix = CUDAArray<unsigned int>(zeroMatrix, QUANTIZED_SIMILARITIES_COUNT, templateDbCount);
	hipMemcpyToSymbol(HIP_SYMBOL(bucketMatrix), &preBucketMatrix, sizeof(CUDAArray<unsigned int>));
	cudaCheckError();

	generateBucketMatrix << <ceilMod(cylinderDbCount, THREADS_PER_BLOCK_MATRIX_GEN), THREADS_PER_BLOCK_MATRIX_GEN >> >(
		LUTSqrt, LUTAngles, LUTPopCountXor);

	hipMemcpyFromSymbol(&preBucketMatrix, HIP_SYMBOL(bucketMatrix), sizeof(CUDAArray<unsigned int>));
	cudaCheckError();

	CUDAArray<float> similaritiesVector = CUDAArray<float>(templateDbCount, 1);
	computeLSS << <ceilMod(templateDbCount, THREADS_PER_BLOCK_LSS), THREADS_PER_BLOCK_LSS >> >
		(LUTTemplateDbLengths, LUTNumPairs, similaritiesVector);


	LUTSqrt.Dispose();
	LUTAngles.Dispose();
	LUTNumPairs.Dispose();
	bucketMatrix.Dispose();

	float* result = similaritiesVector.GetData();

	similaritiesVector.Dispose();

	return result;
}

int main()
{
	unsigned int cylinderCapacity = 1;

	unsigned int *cylinder0Values = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));
	unsigned int *cylinder1Values = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));
	unsigned int *cylinder2Values = (unsigned int *)malloc(cylinderCapacity * sizeof(unsigned int));

	// Test 1
	//memset(cudaCylinder1, 255, cylinderCapacity * sizeof(unsigned int));
	//memset(cudaCylinder2, 255, cylinderCapacity * sizeof(unsigned int));


	// Test 2
	//srand((unsigned int)time(NULL));
	//for (unsigned int i = 0; i < cylinderCapacity; i++) {
	//	cudaCylinder1[i] = rand();
	//	cudaCylinder2[i] = rand();
	//}

	// Test 3 (only for cylinderCapacity == 1)

	cylinder0Values[0] = binToInt("00000000000000000000000000000000");
	cylinder1Values[0] = binToInt("11111111111111111100000000000000");
	cylinder2Values[0] = binToInt("11010001010100001100000000000000");

	Cylinder cylinder0 =
		Cylinder(cylinder0Values, cylinderCapacity, HIP_PI_F / 6, sqrt((float)getOneBitsCountRaw(cylinder0Values, cylinderCapacity)), 0);
	Cylinder cylinder1_0 =
		Cylinder(cylinder1Values, cylinderCapacity, HIP_PI_F / 4, sqrt((float)getOneBitsCountRaw(cylinder1Values, cylinderCapacity)), 0);
	Cylinder cylinder1_1 =
		Cylinder(cylinder1Values, cylinderCapacity, HIP_PI_F / 4, sqrt((float)getOneBitsCountRaw(cylinder1Values, cylinderCapacity)), 1);
	Cylinder cylinder2_1 =
		Cylinder(cylinder2Values, cylinderCapacity, HIP_PI_F / 3, sqrt((float)getOneBitsCountRaw(cylinder2Values, cylinderCapacity)), 1);
	Cylinder cylinder2_2 =
		Cylinder(cylinder2Values, cylinderCapacity, HIP_PI_F / 3, sqrt((float)getOneBitsCountRaw(cylinder2Values, cylinderCapacity)), 2);

	Cylinder db[] = { cylinder1_0, cylinder1_1, cylinder2_1, cylinder2_2, cylinder2_2, cylinder2_2, cylinder2_2 };
	Cylinder query[] = { cylinder2_2 }; // Template index hopefully doesn't matter here
	unsigned int dbTemplateLengths[] = { 1, 2, 4 };
	unsigned int dbTemplateCount = sizeof(dbTemplateLengths) / sizeof(unsigned int);

	float *similarities = getBinTemplateSimilarities(
		query, sizeof(query) / sizeof(Cylinder),
		db, sizeof(db) / sizeof(Cylinder),
		dbTemplateLengths, dbTemplateCount);

	printf("Similarities:\n");
	for (unsigned int i = 0; i < dbTemplateCount; i++)
	{
		printf("%f%s", similarities[i], (i == dbTemplateCount - 1 ? "" : ", "));
	}
	printf("\n");

	// [end] Test 3

	free(cylinder0Values);
	free(cylinder1Values);
	free(cylinder2Values);

	return 0;
}
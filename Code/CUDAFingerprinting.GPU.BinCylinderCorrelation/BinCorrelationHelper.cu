#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "constsmacros.h"
#include "CUDAArray.cuh"
#include <cstring>
#include <time.h>
#include "BinCorrelationHelper.cuh"

void printArray1D(unsigned int* arr, unsigned int length)
{
	for (unsigned int i = 0; i < length; i++) {
		printf("%u ", arr[i]);
	}
	printf("\n");
}

void printCUDAArray1D(CUDAArray<unsigned int> arr)
{
	printf("Print CUDAArray 1D\n");
	printArray1D(arr.GetData(), arr.Width * arr.Height);
	printf("[end] Print CUDAArray 1D\n");
}

void printArray2D(unsigned int* arr, unsigned int width, unsigned int height)
{
	for (unsigned int i = 0; i < height; i++) {
		for (unsigned int j = 0; j < width; j++) {
			printf("%u ", arr[i * width + j]);
		}
		printf("\n");
	}
}

void printCUDAArray2D(CUDAArray<unsigned int> arr)
{
	printf("Print CUDAArray 2D\n");
	printArray2D(arr.GetData(), arr.Width, arr.Height);
	printf("[end] Print CUDAArray 2D\n");
}

__device__ void cudaArrayBitwiseAndDevice(CUDAArray<unsigned int> *fst, CUDAArray<unsigned int> *snd, CUDAArray<unsigned int> *result)
{
	int row = (defaultRow()) % fst->Height;
	int column = (defaultColumn()) % fst->Width;

	if (fst->Width > column && fst->Height > row) {
		unsigned int newValue = fst->At(row, column) & snd->At(row, column);
		result->SetAt(row, column, newValue);
	}
}

__global__ void cudaArrayBitwiseAndGlobal(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd, CUDAArray<unsigned int> result)
{
	if (fst.Width > (defaultColumn()) && fst.Height > (defaultRow())) {
		cudaArrayBitwiseAndDevice(&fst, &snd, &result);
	}
}

CUDAArray<unsigned int> BitwiseAndArray(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd)
{
	dim3 gridSize = dim3(ceilMod(fst.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	CUDAArray<unsigned int> *result = new CUDAArray<unsigned int>(fst.Width, 1);

	cudaArrayBitwiseAndGlobal << <gridSize, blockSize >> >(fst, snd, *result);

	return *result;
}


__device__ void cudaArrayBitwiseXorDevice(CUDAArray<unsigned int> *fst, CUDAArray<unsigned int> *snd, CUDAArray<unsigned int> *result)
{
	int row = (defaultRow()) % fst->Height;
	int column = (defaultColumn()) % fst->Width;

	unsigned int newValue = fst->At(row, column) ^ snd->At(row, column);
	result->SetAt(row, column, newValue);
}

__global__ void cudaArrayBitwiseXorGlobal(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd, CUDAArray<unsigned int> result)
{
	if (fst.Width > (defaultColumn()) && fst.Height > (defaultRow())) {
		cudaArrayBitwiseXorDevice(&fst, &snd, &result);
	}
}


CUDAArray<unsigned int> BitwiseXorArray(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd)
{
	dim3 gridSize = dim3(ceilMod(fst.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	CUDAArray<unsigned int> *result = new CUDAArray<unsigned int>(fst.Width, 1);

	cudaArrayBitwiseXorGlobal << <gridSize, blockSize >> >(fst, snd, *result);

	//printCUDAArray1D(*result);

	return *result;
}

__device__ void cudaArrayWordNormDevice(CUDAArray<unsigned int> *arr, unsigned int* sum)
{
	int row = (defaultRow()) % arr->Height;
	int column = (defaultColumn()) % arr->Width;

	unsigned int x = arr->At(row, column);

	x = __popc(x);

	atomicAdd(sum, x);
}

__global__ void cudaArrayWordNormGlobal(CUDAArray<unsigned int> arr, unsigned int* sum)
{
	if (arr.Width > (defaultColumn()) && arr.Height > (defaultRow())) {
		cudaArrayWordNormDevice(&arr, sum);
	}
}

unsigned int getOneBitsCount(CUDAArray<unsigned int> arr)
{
	dim3 gridSize = dim3(ceilMod(arr.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	unsigned int* sum = (unsigned int*)malloc(sizeof(unsigned int));
	*sum = 0;

	unsigned int* d_sum;
	hipMalloc((unsigned int **)&d_sum, sizeof(unsigned int));
	hipMemcpy(d_sum, sum, sizeof(unsigned int), hipMemcpyHostToDevice);

	cudaArrayWordNormGlobal << <gridSize, blockSize >> >(arr, d_sum);

	hipMemcpy(sum, d_sum, sizeof(unsigned int), hipMemcpyDeviceToHost);

	return *sum;
}

unsigned int getOneBitsCountRaw(unsigned int* arr, unsigned int length)
{
	CUDAArray<unsigned int> cudaArr(arr, length, 1);
	return getOneBitsCount(cudaArr);
}

unsigned int binToInt(char* s)
{
	return (unsigned int)strtoul(s, NULL, 2);
}
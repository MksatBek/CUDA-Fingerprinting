#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "constsmacros.h"
#include "CUDAArray.cuh"
#include <cstring>
#include <time.h>
#include "BinCorrelationHelper.cuh"

void printArray1D(unsigned int* arr, unsigned int length)
{
	for (unsigned int i = 0; i < length; i++) {
		printf("%u ", arr[i]);
	}
	printf("\n");
}

void printCUDAArray1D(CUDAArray<unsigned int> arr)
{
	printf("Print CUDAArray\n");
	printArray1D(arr.GetData(), arr.Width * arr.Height);
	printf("[end] Print CUDAArray\n");
}

__global__ void cudaArrayBitwiseAnd(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd, CUDAArray<unsigned int> result)
{
	int row = defaultRow();
	int column = defaultColumn();

	if (fst.Width > column && fst.Height > row) {
		unsigned int newValue = fst.At(row, column) & snd.At(row, column);
		result.SetAt(row, column, newValue);
	}
}

CUDAArray<unsigned int> BitwiseAndArray(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd)
{
	dim3 gridSize = dim3(ceilMod(fst.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	CUDAArray<unsigned int> *result = new CUDAArray<unsigned int>(fst.Width, 1);

	cudaArrayBitwiseAnd << <gridSize, blockSize >> >(fst, snd, *result);

	return *result;
}


__global__ void cudaArrayBitwiseXor(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd, CUDAArray<unsigned int> result)
{
	int row = defaultRow();
	int column = defaultColumn();

	if (fst.Width > column && fst.Height > row)	{
		unsigned int newValue = fst.At(row, column) ^ snd.At(row, column);
		result.SetAt(row, column, newValue);
	}
}

CUDAArray<unsigned int> BitwiseXorArray(CUDAArray<unsigned int> fst, CUDAArray<unsigned int> snd)
{
	dim3 gridSize = dim3(ceilMod(fst.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	CUDAArray<unsigned int> *result = new CUDAArray<unsigned int>(fst.Width, 1);

	cudaArrayBitwiseXor << <gridSize, blockSize >> >(fst, snd, *result);

	//printCUDAArray1D(*result);

	return *result;
}

__global__ void cudaArrayWordNorm(CUDAArray<unsigned int> arr, unsigned int* sum)
{
	int row = defaultRow();
	int column = defaultColumn();

	if (arr.Width > column && arr.Height > row)	{
		unsigned int x = arr.At(row, column);

		x = x - ((x >> 1) & 0x55555555);
		x = (x & 0x33333333) + ((x >> 2) & 0x33333333);
		x = (x + (x >> 4)) & 0x0F0F0F0F;
		x = x + (x >> 8);
		x = x + (x >> 16);
		x = x & 0x0000003F;

		atomicAdd(sum, x);
	}
}

unsigned int getOneBitsCount(CUDAArray<unsigned int> arr)
{
	dim3 gridSize = dim3(ceilMod(arr.Width, defaultThreadCount), 1, 1);
	dim3 blockSize = dim3(defaultThreadCount, 1, 1);

	unsigned int* sum = (unsigned int*)malloc(sizeof(unsigned int));
	*sum = 0;

	unsigned int* d_sum;
	hipMalloc((unsigned int **)&d_sum, sizeof(unsigned int));
	hipMemcpy(d_sum, sum, sizeof(unsigned int), hipMemcpyHostToDevice);

	cudaArrayWordNorm << <gridSize, blockSize >> >(arr, d_sum);

	hipMemcpy(sum, d_sum, sizeof(unsigned int), hipMemcpyDeviceToHost);

	return *sum;
}



#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "Convolution.cuh"
#include "constsmacros.h"
#include "CUDAArray.cuh"
#include "ImageBinarization.cuh"
#include "ImageLoading.cuh"

int main()
{
	int width = 256;
	int height = 364;
	int *image = loadBmp("2_6.bmp", &width, &height);
	// how to test?
	hipSetDevice(0);
	CUDAArray<int> cudaImg = CUDAArray<int>(image, width, height);
	cudaImg.cudaPtr = image;

	BinarizateImage(cudaImg, 128, image);

	saveBmp("1.bmp", image, width, height);

	hipFree(cudaImg.cudaPtr); // ?

    return 0;
}


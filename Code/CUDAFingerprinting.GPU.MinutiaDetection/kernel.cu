#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <memory.h>
#include <math.h>
#include "constsmacros.h"

//#define DEBUG

#ifdef DEBUG
#include "ImageLoading.cuh"
#endif

struct Minutia
{
	float angle;
	int x;
	int y;
};

#ifndef DEBUG
extern "C"
{
	__declspec(dllexport) int GetMinutias(Minutia* dest, int* data, float* orientation, int width, int height);
}
#endif

#ifdef DEBUG
#include <stdio.h>
#define DBGM(msg) printf("%s\n", msg)
#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0);												 \
   }                                                                 \
}
#else
#define DBGM(msg) ;
#define cudaCheckError() ;
#endif

#define BLOCK_DIM 4

__constant__ int BLACK = 0;
__constant__ int GREY = 128;
__constant__ int WHITE = 255;

__constant__ int w;
__constant__ int h;
/**/
__device__ inline bool IsAvailablePixel(int x, int y)
{
	return !(x < 0 || y < 0 || x >= w || y >= h);
}

__device__ inline int GetPixel(int* data, int x, int y)
{
	return  !IsAvailablePixel(x, y) ?
		WHITE :
		data[((y - blockIdx.y * BLOCK_DIM + 1) % (BLOCK_DIM + 2)) * (BLOCK_DIM + 2) +
			((x - blockIdx.x * BLOCK_DIM + 1) % (BLOCK_DIM + 2))] > GREY ?
				WHITE :
				BLACK;
}

__device__ int MinutiaCode(int* data, int x, int y)
{
	if (GetPixel(data, x, y) != BLACK)
		return -1;
	//check 8-neigbourhood
	bool p[8] = {
		GetPixel(data, x, y - 1) > 0,
		GetPixel(data, x + 1, y - 1) > 0,
		GetPixel(data, x + 1, y) > 0,
		GetPixel(data, x + 1, y + 1) > 0,
		GetPixel(data, x, y + 1) > 0,
		GetPixel(data, x - 1, y + 1) > 0,
		GetPixel(data, x - 1, y) > 0,
		GetPixel(data, x - 1, y - 1) > 0,
	};

	int NeigboursCount = 0;
	for (int i = 1; i < 9; i++)
	{
		NeigboursCount += p[i % 8] ^ p[i - 1] ? 1 : 0;
	}
	NeigboursCount /= 2;
	return NeigboursCount;
}

__device__ inline bool InCircle(int xC, int yC, int R, int x, int y)
{
	return (xC - x) * (xC - x) + (yC - y) * (yC - y) < R * R;
}

__device__ inline float GetCorrectAngle(int* data, float* orientation, int x, int y, int NeigboursCount)
{
	float angle = orientation[(h - 1 - y) * w + x];
	float PI = 3.141592654f;
	//for 'end line' minutia
	if (NeigboursCount == 1)
	{
		if (angle > 0.0f)
		{
			if ((GetPixel(data, x, y - 1) +
				GetPixel(data, x + 1, y - 1) +
				GetPixel(data, x + 1, y))
				<
				(GetPixel(data, x, y + 1) +
				GetPixel(data, x - 1, y + 1) +
				GetPixel(data, x - 1, y)))
			{
				angle += PI;
			}
		}
		else
		{
			if ((GetPixel(data, x, y + 1) +
				GetPixel(data, x + 1, y + 1) +
				GetPixel(data, x + 1, y))
				<
				(GetPixel(data, x, y - 1) +
				GetPixel(data, x - 1, y - 1) +
				GetPixel(data, x - 1, y)))
			{
				angle += PI;
			}
		}
	}
	//for 'fork' minutia
	else if (NeigboursCount == 3)
	{
		for (int r = 1; r < 16; r++)
		{
			float normal = angle + PI / 2;
			int aboveNormal = 0;
			int belowNormal = 0;

			for (int i = -r; i <= r; i++)
			{
				for (int j = -r; j <= r; j++)
				{
					if (i == j && j == 0)
					{
						continue;
					}
					if (GetPixel(data, x + j, y + i) == BLACK &&
						InCircle(x, y, r, x + j, y + i))
					{
						float deltaNormalY = - tan(normal) * j;
						if (((float)i) < deltaNormalY)
						{
							aboveNormal++;
						}
						else
						{
							belowNormal++;
						}
					}
				}
			}
			if (aboveNormal == belowNormal)
			{
				continue;//?
			}
			else
			{
				if ((aboveNormal > belowNormal &&
					tan(angle) > 0.0f) ||
					(aboveNormal < belowNormal &&
					tan(angle) < 0.0f))
				{
					angle += PI;
				}
				break;
			}
		}
	}
	return angle;
}

__device__ inline int GetP(int* data, int x, int y)
{
	return  !IsAvailablePixel(x, y) ?
		WHITE :
		data[(h - 1 - y) * w + x] > GREY ?
			WHITE :
			BLACK;
}

//TODO: separate 
//first kernel: finding minutias
//second kernel: computing angle for every found minutia

__global__ void ProcessPixel(Minutia* dest, int* data, float* orientation)
{
	//x coord of image
	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
	//y coord of image
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;
	
	//caching
	__shared__ int Neigbourhood[(BLOCK_DIM + 2) * (BLOCK_DIM + 2)];

	if (threadIdx.y == 0 && threadIdx.x == 0)
	{
		Neigbourhood[0 * (BLOCK_DIM + 2) + 0] = GetP(data, x - 1, y - 1);
		Neigbourhood[0 * (BLOCK_DIM + 2) + 1] = GetP(data, x, y - 1);
		Neigbourhood[1 * (BLOCK_DIM + 2) + 0] = GetP(data, x - 1, y);
	}
	else if (threadIdx.y == 0 && threadIdx.x == BLOCK_DIM - 1)
	{
		Neigbourhood[0 * (BLOCK_DIM + 2) + (BLOCK_DIM + 1)] = GetP(data, x + 1, y - 1);
		Neigbourhood[0 * (BLOCK_DIM + 2) + (BLOCK_DIM)] = GetP(data, x, y - 1);
		Neigbourhood[1 * (BLOCK_DIM + 2) + (BLOCK_DIM + 1)] = GetP(data, x + 1, y);
	}
	else if (threadIdx.y == BLOCK_DIM - 1 && threadIdx.x == 0)
	{
		Neigbourhood[(BLOCK_DIM + 1) * (BLOCK_DIM + 2) + 0] = GetP(data, x - 1, y + 1);
		Neigbourhood[(BLOCK_DIM)     * (BLOCK_DIM + 2) + 0] = GetP(data, x - 1, y);
		Neigbourhood[(BLOCK_DIM + 1) * (BLOCK_DIM + 2) + 1] = GetP(data, x, y + 1);
	}
	else if (threadIdx.y == BLOCK_DIM - 1 && threadIdx.x == BLOCK_DIM - 1)
	{
		Neigbourhood[(BLOCK_DIM + 1) * (BLOCK_DIM + 2) + (BLOCK_DIM + 1)] = GetP(data, x + 1, y + 1);
		Neigbourhood[(BLOCK_DIM)     * (BLOCK_DIM + 2) + (BLOCK_DIM + 1)] = GetP(data, x + 1, y);
		Neigbourhood[(BLOCK_DIM + 1) * (BLOCK_DIM + 2) + (BLOCK_DIM)]     = GetP(data, x, y + 1);
	}
	else if (threadIdx.y == 0)
	{
		Neigbourhood[0 * (BLOCK_DIM + 2) + threadIdx.x + 1] = GetP(data, x, y - 1);
	}
	else if (threadIdx.x == 0)
	{
		Neigbourhood[(threadIdx.y + 1)* (BLOCK_DIM + 2) + 0] = GetP(data, x - 1, y);
	}
	else if (threadIdx.y == BLOCK_DIM - 1)
	{
		Neigbourhood[(BLOCK_DIM + 1) * (BLOCK_DIM + 2) + threadIdx.x + 1] = GetP(data, x, y + 1);
	}
	else if (threadIdx.x == BLOCK_DIM - 1)
	{
		Neigbourhood[(threadIdx.y + 1)* (BLOCK_DIM + 2) + (BLOCK_DIM + 1)] = GetP(data, x + 1, y);
	}

	Neigbourhood[(threadIdx.y + 1) * (BLOCK_DIM + 2) + threadIdx.x + 1] = GetP(data, x, y);
	
	__syncthreads();
	
	if (!IsAvailablePixel(x, y))
	{
		return;
	}

	int minutiasNumber = y * w + x;

	dest[minutiasNumber].x = -1;
	dest[minutiasNumber].y = -1;
	dest[minutiasNumber].angle = -1.0f;

	int NeigboursCount = MinutiaCode(Neigbourhood, x, y);
	//count == 0 <=> isolated point - NOT minutia
	//count == 1 <=> 'end line' - minutia
	//count == 2 <=> part of the line - NOT minutia
	//count == 3 <=> 'fork' - minutia
	//count >= 3 <=> composit minutia - ignoring in this implementation
	bool IsMinutia = ((NeigboursCount == 1) || (NeigboursCount == 3));

	if (IsMinutia)
	{
		dest[minutiasNumber].x = x;
		dest[minutiasNumber].y = y;
		dest[minutiasNumber].angle = GetCorrectAngle(
			Neigbourhood,
			orientation,
			x,
			y,
			NeigboursCount
		);
	}/* maybe, if it will be here, we can decrease time of an execution?
	else
	{
		dest[minutiasNumber].x = -1;
		dest[minutiasNumber].y = -1;
		dest[minutiasNumber].angle = -1.0f;
	}*/
}

//shift minutias data to beginning of array
int ShrinkResult(Minutia* dest, Minutia* destBuffer, int width, int height)
{
	int minutiasNumber = 0;
	int size = width * height;
	for (int i = 0; i < size; i++)
	{
		if (destBuffer[i].x > -1)
		{
			dest[minutiasNumber].x = destBuffer[i].x;
			dest[minutiasNumber].y = destBuffer[i].y;
			dest[minutiasNumber].angle = destBuffer[i].angle;
			minutiasNumber++;
		}
	}
	free(destBuffer);
	return minutiasNumber;
}

//returns number of found minutias
//in result:
//dest[i * 3 + 0] - x coord of i's minutia
//dest[i * 3 + 1] - y coord of i's minutia
//dest[i * 3 + 2] - direction of i's minutia
int GetMinutias(Minutia* dest, int* data, float* orientation, int width, int height)
{
#ifdef DEBUG
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
#endif

	hipMemcpyToSymbol(HIP_SYMBOL(w), &width, sizeof(width));
	cudaCheckError();

	hipMemcpyToSymbol(HIP_SYMBOL(h), &height, sizeof(height));
	cudaCheckError();

	Minutia* destBuffer = (Minutia*)malloc(sizeof(Minutia) * height * width);

	//allocate memory on device & initialize
	Minutia* devDest;
	hipMalloc((void**)&devDest, sizeof(Minutia) * height * width);
	cudaCheckError();

	int* devData;
	hipMalloc((void**)&devData, sizeof(int) * height * width);
	cudaCheckError();
	hipMemcpy(devData, data, sizeof(int) * height * width, hipMemcpyHostToDevice);
	cudaCheckError();

	float* devOrientation;
	hipMalloc((void**)&devOrientation, sizeof(float) * height * width);
	cudaCheckError();
	hipMemcpy(devOrientation, orientation, sizeof(float) * height * width, hipMemcpyHostToDevice);
	cudaCheckError();

	int blocksRowSize = ceilMod(width, BLOCK_DIM);
	int blocksColumnSize = ceilMod(height, BLOCK_DIM);
	dim3 gridSize = dim3(blocksRowSize, blocksColumnSize);
	dim3 blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);

	ProcessPixel <<<gridSize, blockSize>>>(devDest, devData, devOrientation);

	//getting results & free device memory
	hipMemcpy(destBuffer, devDest, sizeof(Minutia) * height * width, hipMemcpyDeviceToHost);
	cudaCheckError();
	hipFree(devDest);
	cudaCheckError(); 
	hipFree(devData);
	cudaCheckError();
	hipFree(devOrientation);
	cudaCheckError();

#ifdef DEBUG
	hipEventRecord(stop, 0);	
	hipEventSynchronize(stop); 
	float time;	
	hipEventElapsedTime(&time, start, stop); 

	printf("\nGetMinutias without ShrinkResult execution time: %e ms", time);
#endif
	int minutiasCount = ShrinkResult(dest, destBuffer, width, height);


	return minutiasCount;
}

#ifdef DEBUG
void prntArr(Minutia arr[], int size)
{
	while (-1 < --size)
	{
		printf("X=%3.d, Y=%3.d, ANGLE=%e     ", arr[size].x, arr[size].y, arr[size].angle);
		if (size % 2 == 0)
		{
			printf("\n");
		}
	}
}

void initOr(float* or, int size)
{
	while (-1 < --size)
	{
		or[size] = (float)size;
	}
}

int* overlapMinutias(int* img, Minutia ms[], int minSize, int width, int height)
{
	while (-1 < --minSize)
	{
		img[(height - 1 - ms[minSize].y) * width + ms[minSize].x] = 128;
	}
	return img;
}

int main()
{
	hipSetDevice(0);
	int width = 0;
	int height = 0;
	int* img = loadBmp("D:\\Ucheba\\Programming\\summerSchool\\Code\\Debug\\skeleton.bmp", &width, &height);//test file from folder with executable file
	
	float* orientation = (float*)malloc(sizeof(float) * width * height);
	initOr(orientation, width * height);
	
	Minutia* minutiasArray = (Minutia*)malloc(sizeof(Minutia) * width * height);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	int minutiasCount = GetMinutias(
		minutiasArray,
		img,
		orientation,
		width,
		height
	);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);

	printf("\nTotal GetMinutias execution time:                %e ms\nBLOCK DIM: %d\nMinutias found: %d\n", time, BLOCK_DIM, minutiasCount);
	
	prntArr(minutiasArray, minutiasCount);

	overlapMinutias(img, minutiasArray, minutiasCount, width, height);
	saveBmp("D:\\Ucheba\\Programming\\summerSchool\\Code\\Debug\\resultCUDA.bmp", img, width, height);
	system("D:\\Ucheba\\Programming\\summerSchool\\Code\\Debug\\resultCUDA.bmp");

	free(img);
	free(orientation);
	free(minutiasArray);
	return 0;
}
#endif
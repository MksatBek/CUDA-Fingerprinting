#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES 
#include "hip/hip_runtime.h"
#include ""
#include "CUDAArray.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include "ImageLoading.cuh"
#include "OrientationField.cuh"
#include "hip/hip_math_constants.h"
#include "ImageEnhancement.cuh"
#include "Filter.cuh"
#include "Convolution.cuh"
const int l = 32;

extern "C"
{
	__declspec(dllexport) void GetFrequency(float* res, float* image, int height, int width, float* orientMatrix, int interpolationFilterSize = 7,
		int interpolationSigma = 1, int lowPassFilterSize = 25, int lowPassFilterSigma = 4, int w = 16);
}

__device__ void CalculateSignatureLine(float* res, CUDAArray<float>* img, int width, int height, int index, float angle, int x, int y, int w)
{
	float angleSin = sin(angle);
	float angleCos = cos(angle);

	//int iStart = y - (3 * w / 2 -1);
	//int jStart = x - (3 * w / 2 -1);

	float signature = 0;
	for (int d = 0; d < w; d++)
	{
		int indX = (int)(x + (d - w / 2) * angleCos + (index - l / 2) * angleSin);
		int indY = (int)(y + (d - w / 2) * angleSin + (l / 2 - index) * angleCos);
		if ((indX < 0) || (indY < 0) || (indX >= width) || (indY >= height))
			continue;
	/*	if ((indY - iStart < 0) || (indY - iStart >= 48) || (indX - jStart < 0) || (indX - jStart >= 48))
		{
			signature++;
		}*/
		
		signature += img->At(indX, indY);
			////signature += buffer[1];
			//signature += 1;
	}
	signature /= w;

	*res = signature;
}

__global__ void CalculateFrequencyPixel(CUDAArray<float> res, CUDAArray<float> img, CUDAArray<float> orientMatr, int w)
{
	int column = defaultColumn();
	int row = defaultRow();
	if ((column < img.Width) && (row < img.Height)) {

		int prevMin = -1;
		int lengthsSum = 0;
		int summandNum = 0;

		/*__shared__ float buffer[48][48];
		__shared__ int iStart, iFinish, jStart, jFinish;*/
		//center of the block of even size shall be the left upper pixel of the four central ones.
	/*	int check1 = (row - (w / 2 - 1)) % w;
		int check2 = (column - (w / 2 - 1)) % w;
		if (row == 7 && column == 7)
		{
			summandNum++;
		}*/
		//if (((row - (w / 2 - 1)) % w == 0) && ((column - (w / 2 - 1)) % w == 0))
		//{
		//	iStart = row - ((3 * w) / 2 - 1);
		//	iFinish = row + (3 * w) / 2 + 1;
		//	if (iStart < 0) iStart = 0;
		//	if (iFinish > img.Height) iFinish = img.Height;

		//	jStart = column - ((3 * w) / 2 - 1);
		//	jFinish = column + (3 * w) / 2 + 1;
		//	//if (iStart == -1)
		//	//{
		//	//	jStart--;
		//	//}
		//	if (jStart < 0) jStart = 0;
		//	if (jFinish > img.Width) jFinish = img.Width;
		//	for (int i = iStart; i < iFinish; i++)
		//	for (int j = jStart; j < jFinish; j++)
		//	{
		//		//if (((i - iStart) * (3 * w / 2) + (j - jStart) <= 9 * w * w - 1) && ((i - iStart) * (3 * w / 2) + (j - jStart) >= 0))
		//		//{
		//			buffer[(i - iStart)][(j - jStart)] = img.At(i, j);
		//		//}
		//		//buffer[(i - iStart) * (3 * w / 2) + (j - jStart)] = img.At(i, j);
		//		//lengthsSum = img.At(i, j);
		//	}
		//}
		//__syncthreads();
		float a, b, c;
		CalculateSignatureLine(&a, &img, img.Width, img.Height, 0, orientMatr.At(row, column), column, row, w);
		CalculateSignatureLine(&b, &img, img.Width, img.Height, 1, orientMatr.At(row, column), column, row, w);
		for (int i = 1; i < l - 1; i++)
		{
			CalculateSignatureLine(&c, &img, img.Width, img.Height, i + 1, orientMatr.At(row, column), column, row, w);
			//In comparison below there has to be non-zero value so that we would be able to ignore minor irrelevant pits of black.
			if ((a - b > 0.5) && (c - b  > 0.5))
			{
				if (prevMin != -1)
				{
					lengthsSum += i - prevMin;
					summandNum++;
					prevMin = i;
				}
				else
				{
					prevMin = i;
				//	lengthsSum--;
				}
			}
			a = b;
			b = c;
		}
		float frequency = (float)summandNum / lengthsSum;
		if ((lengthsSum <= 0) || (frequency > 1.0f / 3.0f) || (frequency < 0.04f))
			frequency = -1;
		res.SetAt(row, column, frequency);
	}
}

void CalculateFrequency(float* res, float* image, int height, int width, float* orientMatrix, int w)
{
	CUDAArray<float> img               = CUDAArray<float>(image, width, height);
	CUDAArray<float> frequencyMatrix   = CUDAArray<float>(width, height);
	CUDAArray<float> orientationMatrix = CUDAArray<float>(orientMatrix, width, height);

	/*dim3 blockSize = dim3 (defaultThreadCount/2, defaultThreadCount/2);
	dim3 gridSize = dim3(ceilMod(width, (defaultThreadCount/2)), ceilMod(height, (defaultThreadCount/2)));*/

	dim3 blockSize = dim3(16, 16);
	dim3 gridSize = dim3(ceilMod(width, 16), ceilMod(height, 16));
	CalculateFrequencyPixel<<<gridSize, blockSize>>>(frequencyMatrix, img, orientationMatrix, w);
	frequencyMatrix.GetData(res);

	img.Dispose();
	frequencyMatrix.Dispose();
	orientationMatrix.Dispose();
}

__global__ void InterpolatePixel(CUDAArray<float> frequencyMatrix, CUDAArray<float> result, bool* needMoreInterpolationFlag, CUDAArray<float> filter, int w)
{
	int row    = defaultRow();
	int column = defaultColumn();

	int height = frequencyMatrix.Height;
	int width  = frequencyMatrix.Width;

	*needMoreInterpolationFlag = false;

	if (row < height && column < width) {
		if (frequencyMatrix.At(row, column) == -1.0)
		{
			int center = filter.Width / 2;
			int upperCenter = (filter.Width & 1) == 0 ? center - 1 : center;

			float numerator = 0;
			float denominator = 0;
			for (int drow = -upperCenter; drow <= center; drow++)
			{
				for (int dcolumn = -upperCenter; dcolumn <= center; dcolumn++)
				{
					float filterValue = filter.At(center - drow, center - dcolumn);
					int indexRow = row + drow * w;
					int indexColumn = column + dcolumn * w;

					if (indexRow < 0)    indexRow = 0;
					if (indexColumn < 0) indexColumn = 0;
					if (indexRow >= height)   indexRow = height - 1;
					if (indexColumn >= width) indexColumn = width - 1;

					float freqVal = frequencyMatrix.At(indexRow, indexColumn);
					//Mu:
					float freqNumerator = freqVal;
					if (freqNumerator <= 0) freqNumerator = 0;
					//Delta:
					float freqDenominator = freqVal;
					if (freqDenominator + 1 <= 0) freqDenominator = 0;
					else freqDenominator = 1;

					numerator += filterValue * freqNumerator;
					denominator += filterValue * freqDenominator;
				}
			}
			float freqBuf = numerator / denominator;
			if (freqBuf != freqBuf || freqBuf > 1.0 / 3.0 || freqBuf < 0.04)
			{
				freqBuf = -1;
				*needMoreInterpolationFlag = true;
			}
			result.SetAt(row, column, freqBuf);
		}
		else
			result.SetAt(row, column, frequencyMatrix.At(row, column));
	}
}

void Interpolate(int imgWidth, int imgHeight, float* res, bool* needMoreInterpolationFlag, float* frequencyMatr, int filterSize, float sigma, int w)
{
	CUDAArray<float> result = CUDAArray<float>(imgWidth, imgHeight);
	CUDAArray<float> frequencyMatrix = CUDAArray<float>(frequencyMatr, imgWidth, imgHeight);

	CUDAArray<float> filter = CUDAArray<float>(MakeGaussianFilter(filterSize, sigma), filterSize, filterSize);

	bool* dev_needMoreInterpolationFlag;
	hipMalloc((void**)&dev_needMoreInterpolationFlag, sizeof(bool));

	dim3 blockSize = dim3(defaultThreadCount, defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(imgWidth, defaultThreadCount), ceilMod(imgHeight, defaultThreadCount));
	InterpolatePixel << <gridSize, blockSize >> >(frequencyMatrix, result, dev_needMoreInterpolationFlag,filter, w);

	hipMemcpy(needMoreInterpolationFlag, dev_needMoreInterpolationFlag, sizeof(bool), hipMemcpyDeviceToHost);
	if (needMoreInterpolationFlag)
		Interpolate(imgWidth, imgHeight, res, needMoreInterpolationFlag, result.GetData(), filterSize, sigma, w);

	result.GetData(res);

	hipFree(dev_needMoreInterpolationFlag);
	result.Dispose();
	frequencyMatrix.Dispose();
	filter.Dispose();
}

void FilterFrequencies(int imgWidth, int imgHeight, float* res, float* frequencyMatr, int filterSize, float sigma, int w)
{
	CUDAArray<float> result = CUDAArray<float>(imgWidth, imgHeight);
	CUDAArray<float> frequencyMatrix = CUDAArray<float>(frequencyMatr, imgWidth, imgHeight);

	CUDAArray<float> lowPassFilter = CUDAArray<float>(MakeGaussianFilter(filterSize, sigma), filterSize, filterSize);

	Convolve(result, frequencyMatrix, lowPassFilter, w);
	result.GetData(res);

	result.Dispose();
	frequencyMatrix.Dispose();
	lowPassFilter.Dispose();
}

void GetFrequency(float* res, float* image, int height, int width, float* orientMatrix, int interpolationFilterSize, 
	int interpolationSigma, int lowPassFilterSize, int lowPassFilterSigma, int w)
{
	float* initialFreq = (float*) malloc(height * width * sizeof(float));
	CalculateFrequency(initialFreq, image, height, width, orientMatrix, w);

	float* interpolatedFreq = (float*)malloc(height * width * sizeof(float));
	Interpolate(width, height, interpolatedFreq, false, initialFreq, interpolationFilterSize, interpolationSigma, w);

	FilterFrequencies(width, height, res, interpolatedFreq, lowPassFilterSize, lowPassFilterSigma, w);

	free(initialFreq);
	free(interpolatedFreq);
}

void main()
{
	hipSetDevice(0);
	int width;
	int height;
	int w = 16;
	char* filename = "..//4_8.bmp";  //Write your way to bmp file
	int* img = loadBmp(filename, &width, &height);
	float* source = (float*)malloc(height*width*sizeof(float));
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
			source[i * width + j] = (float)img[i * width + j];
		}
	//float* b = (float*)malloc(height * width * sizeof(float));
	//float* c = (float*)malloc(height * width * sizeof(float));
	//float* d = (float*)malloc(height * width * sizeof(float));
	float* g = (float*)malloc(height * width * sizeof(float));
	float* h = (float*)malloc(height * width * sizeof(float));
	float* orMatr =	OrientationFieldInPixels(source, width, height);

	GetFrequency(g, source, height, width, orMatr, 7, 1, 25, 4, 16);
	
	//CalculateFrequency(b, source, height, width, orMatr, w);
	//Interpolate(width, height, d, false, b, 7, 1, 16);
	//
	//FilterFrequencies(width, height, b, d, 19, 3, 16);
	//Enhance(source, width, height, c, orMatr, b, 32, 8);
	Enhance(source, width, height, h, orMatr, g, 32, 8);
//	saveBmp("..\\res.bmp", c, width, height);
	saveBmp("..\\res.bmp", h, width, height);
	free(source);
	free(img);
	free(g);
	free(h);
	hipDeviceReset();
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include "constsmacros.h"
#include <stdlib.h>
#include "CUDAArray.cuh"
#include <float.h>
#include "hip/hip_math_constants.h"
#include "math_functions.h"
#include "CUDAArray.cuh"
#include <math.h>
#include "Filter.cuh"

__global__ void cudaCreateGaborFilter(CUDAArray<float> filters, int size, float frequency, float bAngle)
{
	float aCos = cos(/*HIP_PI_F / 2 +*/ bAngle * (blockIdx.x));
	float aSin = sin(/*HIP_PI_F / 2 +*/ bAngle * (blockIdx.x));

	int center = size / 2;

	int dX = threadIdx.x - center;
	int dY = threadIdx.y - center;

	float xDash = dX * aSin + dY * aCos;
	float yDash = -dX *aCos + dY * aSin;
	float cellExp = exp(-0.5 * (xDash * xDash / 16 + yDash * yDash / 16));
	float cellCos = cos(2 * HIP_PI_F * xDash * frequency);

	filters.SetAt(threadIdx.x, blockDim.x * blockIdx.x + threadIdx.y, cellExp * cellCos);
}

CUDAArray<float> MakeGabor16Filters(int angleNum, float frequency)
{
	CUDAArray<float> filters = CUDAArray<float>(16 * angleNum, 16);
	
	float bAngle = (float) HIP_PI_F / angleNum;

	cudaCreateGaborFilter << < dim3(angleNum), dim3(16, 16) >> > (filters, 16, frequency, bAngle);

	return filters;
}

CUDAArray<float> MakeGabor32Filters(int angleNum, float frequency)
{
	CUDAArray<float> filters = CUDAArray<float>(32 * angleNum, 32);

	float bAngle = (float)HIP_PI_F / angleNum;

	cudaCreateGaborFilter << < dim3(angleNum), dim3(32, 32) >> > (filters, 32, frequency, bAngle);

	return filters;
}

//int main()
//{
//	float* b = (float*)malloc(16*16*8*sizeof(float));
//
//	MakeGabor16Filters(b, 8, (float) 1 / 9);
//
//	return 0;
//}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include "constsmacros.h"
#include <stdlib.h>
#include "CUDAArray.cuh"
#include <float.h>
#include "hip/hip_math_constants.h"
#include "math_functions.h"
#include "CUDAArray.cuh"

extern "C"
{
	__declspec(dllexport) void MakeGabor16Filters(float* filter, int angleNum, float frequency);
	__declspec(dllexport) void MakeGabor32Filters(float* filter, int angleNum, float frequency);
}

__global__ void cudaCreateGaborFilter(CUDAArray<float> filters, int size, float frequency, float bAngle)
{
	float aCos = cos(HIP_PI_F / 2 + bAngle * (blockIdx.x));
	float aSin = sin(HIP_PI_F / 2 + bAngle * (blockIdx.x));

	int center = size / 2;
	int upperCenter = (size & 1) == 0 ? center - 1 : center;

	int dX = threadIdx.x - center;
	int dY = threadIdx.y - center;

	float xDash = dX * aSin + dY * aCos;
	float yDash = -dX *aCos + dY * aSin;
	float cellExp = exp(-0.5 * (xDash * xDash / 16 + yDash * yDash / 16));
	float cellCos = cos(2 * HIP_PI_F * xDash * frequency);

	filters.SetAt(threadIdx.x, blockDim.x * blockIdx.x + threadIdx.y, cellExp * cellCos);
}

void MakeGabor16Filters(float* filter, int angleNum, float frequency)
{
	CUDAArray<float> filters = CUDAArray<float>(16 * angleNum, 16);
	
	float bAngle = (float) HIP_PI_F / angleNum;

	cudaCreateGaborFilter << < dim3(angleNum), dim3(16, 16) >> > (filters, 16, frequency, bAngle);

	filters.GetData(filter);
}

void MakeGabor32Filters(float* filter, int angleNum, float frequency)
{
	CUDAArray<float> filters = CUDAArray<float>(32 * angleNum, 32);

	float bAngle = (float)HIP_PI_F / angleNum;

	cudaCreateGaborFilter << < dim3(angleNum), dim3(32, 32) >> > (filters, 32, frequency, bAngle);

	filters.GetData(filter);
}

//int main()
//{
//	float* b = (float*)malloc(16*16*8*sizeof(float));
//
//	MakeGabor16Filters(b, 8, (float) 1 / 9);
//
//	return 0;
//}
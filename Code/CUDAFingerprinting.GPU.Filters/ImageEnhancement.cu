#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES 
#include "hip/hip_runtime.h"
#include ""
#include "CUDAArray.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include "Filter.cuh"
#include "ImageLoading.cuh"
#include "OrientationField.cuh"
#include "hip/hip_math_constants.h"
extern "C"
{
	__declspec(dllexport) void Enhance(float* source, int imgWidth, int imgHeight, float* res, float* orientationMatrix,
		float frequency, int filterSize, int angleNum);
}
__global__ void EnhancePixel(CUDAArray<float> img, CUDAArray<float> result, CUDAArray<float> orientMatrix, float frequency,
	CUDAArray<float> filters, int angleNum, float* angles)
{
	int row = defaultRow();
	int column = defaultColumn();

	int filterSize  = filters.Height;
	int center      = filterSize / 2;
	int upperCenter = (filterSize & 1) == 0 ? center - 1 : center;

	if (row < img.Height && column < img.Width) {
		float diff = FLT_MAX;
		int angle = 0;
		for (int angInd = 0; angInd < angleNum; angInd++)
		{
			if (abs(angles[angInd] - orientMatrix.At(row, column)) < diff)
			{
				angle = angInd;
				diff = abs(angles[angInd] - orientMatrix.At(row, column));
			}
		}

		float sum = 0;
		for (int drow = -upperCenter; drow <= center; drow++)
		{
			for (int dcolumn = -upperCenter; dcolumn <= center; dcolumn++)
			{
				float filterValue = filters.At(center - drow, filterSize* angle + (center - dcolumn));
				
				int indexRow = row + drow;
				int indexColumn = column + dcolumn;

				if (indexRow < 0)    indexRow    = 0;
				if (indexColumn < 0) indexColumn = 0;
				if (indexRow >= img.Height)   indexRow    = img.Height - 1;
				if (indexColumn >= img.Width) indexColumn = img.Width - 1;

				sum += filterValue * img.At(indexRow, indexColumn);
			}
		}
		//sum = (((int)sum) % 256 + (sum - ((int)sum)));//I would've written 'sum %= 256' if 'sum' was integer.
		if (sum < 0) sum = 0;
		result.SetAt(row, column, sum);
	}
}

void Enhance(float* source, int imgWidth, int imgHeight, float* res, float* orientationMatrix, 
	float frequency, int filterSize, int angleNum)
{
	CUDAArray<float> result       = CUDAArray<float>(imgWidth, imgHeight);
	CUDAArray<float> img          = CUDAArray<float>(source, imgWidth, imgHeight);
	CUDAArray<float> orientMatrix = CUDAArray<float>(orientationMatrix, imgWidth, imgHeight);

	float* angles = (float*)malloc(angleNum * sizeof(float));//passing small array is better than creating it multiple times, I think.
	const float constAngle = HIP_PI_F / angleNum;
	for (int i = 0; i < angleNum; i++)
		angles[i] = constAngle * i - HIP_PI_F / 2;
	float* dev_angles;
	hipMalloc((void**)&dev_angles, angleNum * sizeof(float));
	hipMemcpy(dev_angles, angles, angleNum * sizeof(float), hipMemcpyHostToDevice);
	
	CUDAArray<float> filters = MakeGabor32Filters(angleNum, frequency);

	dim3 blockSize = dim3(defaultThreadCount, defaultThreadCount);
	dim3 gridSize  = dim3(ceilMod(imgWidth, defaultThreadCount), ceilMod(imgHeight, defaultThreadCount));
	EnhancePixel << <gridSize, blockSize >> >(img, result, orientMatrix, frequency, filters, angleNum, dev_angles);
	result.GetData(res);
}

void main()
{
	int width;
	int height;
	char* filename = "..\\4_8.bmp";  //Write your way to bmp file
	int* img = loadBmp(filename, &width, &height);
	float* source = (float*)malloc(height*width*sizeof(float));
	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
		{
			source[i * width + j] = (float)img[i * width + j];
		}
	float* b = (float*)malloc(height * width * sizeof(float));
	float* orMatr = OrientationFieldInPixels(source, width, height);
	Enhance(source, width, height, b, orMatr, (float)1 / 9, 32, 8);
	saveBmp("..\\res.bmp", b, width, height);

	free(source);
	free(img);
	free(b);
}

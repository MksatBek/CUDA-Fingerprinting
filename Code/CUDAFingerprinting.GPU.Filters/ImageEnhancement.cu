#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES 
#include "hip/hip_runtime.h"
#include ""
#include "CUDAArray.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include "Filter.cuh"
#include "ImageLoading.cuh"
#include "OrientationField.cuh"
#include "hip/hip_math_constants.h"
#include "ImageEnhancement.cuh"
__global__ void EnhancePixel(CUDAArray<float> img, CUDAArray<float> result, CUDAArray<float> orientMatrix, CUDAArray<float> frequencyMatrix,
	CUDAArray<float> filters, int filterSize, int angleNum, float* angles)
{
	int row = defaultRow();
	int column = defaultColumn();

	int center      = filterSize / 2;
	int upperCenter = (filterSize & 1) == 0 ? center - 1 : center;

	const int freqNum = 4;
	float frArr[freqNum] = {
		1.0 / 25.0,
		1.0 / 16.0,
		1.0 / 9.0,
		1.0 / 3.0
	};

	if (row < img.Height && column < img.Width) {
		float diff = FLT_MAX;
		int angle = 0;
		for (int angInd = 0; angInd < angleNum; angInd++)
		{
			if (abs(angles[angInd] - orientMatrix.At(row, column)) < diff)
			{
				angle = angInd;
				diff = abs(angles[angInd] - orientMatrix.At(row, column));
			}
		}

		diff = FLT_MAX;
		int freq = 0;
		for (int freqInd = 0; freqInd < freqNum; freqInd++)
			if (abs(frArr[freqInd] - frequencyMatrix.At(row, column)) < diff)
			{
				freq = freqInd;
				diff = abs(frArr[freqInd] - frequencyMatrix.At(row, column));
			}

		float sum = 0;
		for (int drow = -upperCenter; drow <= center; drow++)
		{
			for (int dcolumn = -upperCenter; dcolumn <= center; dcolumn++)
			{
				float filterValue = filters.At(filterSize * freq + (center - drow), filterSize* angle + (center - dcolumn));
				int indexRow = row + drow;
				int indexColumn = column + dcolumn;

				if (indexRow < 0)    indexRow    = 0;
				if (indexColumn < 0) indexColumn = 0;
				if (indexRow >= img.Height)   indexRow    = img.Height - 1;
				if (indexColumn >= img.Width) indexColumn = img.Width - 1;

				sum += filterValue * img.At(indexRow, indexColumn);
			}
		}
		if (sum > 255) sum = 255;
		if (sum < 0)   sum = 0;
		result.SetAt(row, column, sum);
	}
}
//For filters of arbitrary size. Works only for filters with size less or equal to 32.
void Enhance(float* source, int imgWidth, int imgHeight, float* res, float* orientationMatrix, 
	float* frequencyMatr, int filterSize, int angleNum)
{
	CUDAArray<float> result       = CUDAArray<float>(imgWidth, imgHeight);
	CUDAArray<float> img          = CUDAArray<float>(source, imgWidth, imgHeight);
	CUDAArray<float> orientMatrix = CUDAArray<float>(orientationMatrix, imgWidth, imgHeight);
	CUDAArray<float> frequencyMatrix = CUDAArray<float>(frequencyMatr, imgWidth, imgHeight);

	const int freqNum = 4;
	float frArr[freqNum] = {
		1.0 / 25.0,
		1.0 / 16.0,
		1.0 / 9.0,
		1.0 / 3.0
	};

	float* angles = (float*)malloc(angleNum * sizeof(float));//passing small array is better than creating it multiple times, I think.
	const float constAngle = HIP_PI_F / angleNum;
	for (int i = 0; i < angleNum; i++)
		angles[i] = constAngle * i - HIP_PI_F / 2;
	float* dev_angles;
	hipMalloc((void**)&dev_angles, angleNum * sizeof(float));
	hipMemcpy(dev_angles, angles, angleNum * sizeof(float), hipMemcpyHostToDevice);

	CUDAArray<float> filters = MakeGaborFilters(filterSize, angleNum, frArr, freqNum);

	dim3 blockSize = dim3(defaultThreadCount, defaultThreadCount);
	dim3 gridSize  = dim3(ceilMod(imgWidth, defaultThreadCount), ceilMod(imgHeight, defaultThreadCount));
	EnhancePixel << <gridSize, blockSize >> >(img, result, orientMatrix, frequencyMatrix, filters, filterSize, angleNum, dev_angles);
	result.GetData(res);
}

//For filters 32x32.
void Enhance32(float* source, int imgWidth, int imgHeight, float* res, float* orientationMatrix,
	float* frequencyMatr, int angleNum)
{
	CUDAArray<float> result = CUDAArray<float>(imgWidth, imgHeight);
	CUDAArray<float> img = CUDAArray<float>(source, imgWidth, imgHeight);
	CUDAArray<float> orientMatrix = CUDAArray<float>(orientationMatrix, imgWidth, imgHeight);
	CUDAArray<float> frequencyMatrix = CUDAArray<float>(frequencyMatr, imgWidth, imgHeight);

	const int freqNum = 4;
	float frArr[freqNum] = {
		1.0 / 25.0,
		1.0 / 16.0,
		1.0 / 9.0,
		1.0 / 3.0
	};

	float* angles = (float*)malloc(angleNum * sizeof(float));//passing small array is better than creating it multiple times, I think.
	const float constAngle = HIP_PI_F / angleNum;
	for (int i = 0; i < angleNum; i++)
		angles[i] = constAngle * i - HIP_PI_F / 2;
	float* dev_angles;
	hipMalloc((void**)&dev_angles, angleNum * sizeof(float));
	hipMemcpy(dev_angles, angles, angleNum * sizeof(float), hipMemcpyHostToDevice);

	CUDAArray<float> filters = MakeGabor32Filters(angleNum, frArr, freqNum);

	dim3 blockSize = dim3(defaultThreadCount, defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(imgWidth, defaultThreadCount), ceilMod(imgHeight, defaultThreadCount));
	EnhancePixel << <gridSize, blockSize >> >(img, result, orientMatrix, frequencyMatrix, filters, 32, angleNum, dev_angles);
	result.GetData(res);

}

//For filters 16x16.
void Enhance16(float* source, int imgWidth, int imgHeight, float* res, float* orientationMatrix,
	float* frequencyMatr, int angleNum)
{
	CUDAArray<float> result = CUDAArray<float>(imgWidth, imgHeight);
	CUDAArray<float> img = CUDAArray<float>(source, imgWidth, imgHeight);
	CUDAArray<float> orientMatrix = CUDAArray<float>(orientationMatrix, imgWidth, imgHeight);
	CUDAArray<float> frequencyMatrix = CUDAArray<float>(frequencyMatr, imgWidth, imgHeight);

	const int freqNum = 4;
	float frArr[freqNum] = {
		1.0 / 25.0,
		1.0 / 16.0,
		1.0 / 9.0,
		1.0 / 3.0
	};

	float* angles = (float*)malloc(angleNum * sizeof(float));//passing small array is better than creating it multiple times, I think.
	const float constAngle = HIP_PI_F / angleNum;
	for (int i = 0; i < angleNum; i++)
		angles[i] = constAngle * i - HIP_PI_F / 2;
	float* dev_angles;
	hipMalloc((void**)&dev_angles, angleNum * sizeof(float));
	hipMemcpy(dev_angles, angles, angleNum * sizeof(float), hipMemcpyHostToDevice);

	CUDAArray<float> filters = MakeGabor16Filters(angleNum, frArr, freqNum);

	dim3 blockSize = dim3(defaultThreadCount, defaultThreadCount);
	dim3 gridSize = dim3(ceilMod(imgWidth, defaultThreadCount), ceilMod(imgHeight, defaultThreadCount));
	EnhancePixel << <gridSize, blockSize >> >(img, result, orientMatrix, frequencyMatrix, filters, 16, angleNum, dev_angles);
	result.GetData(res);
}
//void main()
//{
//	int width;
//	int height;
//	char* filename = "..\\4_8.bmp";  //Write your way to bmp file
//	int* img = loadBmp(filename, &width, &height);
//	float* source = (float*)malloc(height*width*sizeof(float));
//	for (int i = 0; i < height; i++)
//		for (int j = 0; j < width; j++)
//		{
//			source[i * width + j] = (float)img[i * width + j];
//		}
//	float* b = (float*)malloc(height * width * sizeof(float));
//	float* orMatr = OrientationFieldInPixels(source, width, height);
//	Enhance(source, width, height, b, orMatr, (float)1 / 9, 32, 8);
//	saveBmp("..\\res.bmp", b, width, height);
//
//	free(source);
//	free(img);
//	free(b);
//}

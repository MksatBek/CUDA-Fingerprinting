#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h> 

#include "hip/hip_runtime.h"
#include ""
#include "MinutiaHelper.cuh"

__device__ float sqrLength(Minutia m1, Minutia m2)
{
	return (float)((m1.x - m2.x)*(m1.x - m2.x) + (m1.y - m2.y)*(m1.y - m2.y));
}

__device__ float normalizeAngle(float angle)
{
	angle -= (float)(floor(angle / (2 * M_PI)) * 2 * M_PI);

	return angle;
}

void fingerRead(char *dbPath, int dbSize, Minutia **mins, int *minutiaNum)
{
	FILE *finger;
	char filePath[FILEPATH_LENGTH];
	char fileNum[FILENAME_LENGTH];
	int i, j;

	for (i = 0; i < dbSize; i++)
	{
		filePath[0] = '\0';
		itoa(i, fileNum, 10);
		strncat(filePath, dbPath, FILEPATH_LENGTH);
		strncat(filePath, "\\", FILENAME_LENGTH);
		strncat(filePath, fileNum, FILENAME_LENGTH);
		strncat(filePath, ".txt", FILENAME_LENGTH);

		finger = fopen(filePath, "r");

		fscanf(finger, "%d", &(minutiaNum[i]));
		for (j = 0; j < minutiaNum[i]; j++)
		{
			fscanf(finger, "%d %d %f", &(mins[i][j].x), &(mins[i][j].y), &(mins[i][j].angle));
		}

		fclose(finger);
		/*
		for (j = 0; j < minutiaNum[i]; j++)
		{
			printf("%d %d %f\n", mins[i][j].x, mins[i][j].y, mins[i][j].angle);
		}
		printf("__\n");*/
	}
}

__device__ void cudaReductionSum(float* a, int i, int x)
{
	i /= 2;

	while (i != 0)
	{
		if (x < i)
		{
			a[x] = a[x + i];
		}

		__syncthreads();
		i /= 2;
	}
}

__device__ void cudaReductionSum2D(float* a, int i, int j, int x, int y)
{
	j /= i*j;

	while (j != 0)
	{
		if (x*i + y < j)
		{
			*((a + x*i) + y) = *((a + x*i) + y + j);
		}

		__syncthreads();
		i /= 2;
	}
}

//__device__ void cudaSumBlock()
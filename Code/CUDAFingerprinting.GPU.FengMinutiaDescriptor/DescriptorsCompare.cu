#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "MinutiaHelper.cuh"
/*
//#include ""

__device__ void transformate(Minutia* src, Minutia center, Minutia* dst, int j)
{
	float angle = center.angle - src[j].angle;
	float cosAngle = cos(angle);
	float sinAngle = sin(angle);
	float dx = src[j].x - center.x;
	float dy = src[j].y - center.y;

	int x = (int)round(dx * cosAngle + dy * sinAngle) + center.x;
	int y = (int)round(-dx * sinAngle + dy * cosAngle) + center.y;

	Minutia temp(normalizeAngle(src[j].angle + angle), x, y);

	dst[j] = temp;
}

__device__ void matchingPoints(Descriptor desc1, Descriptor desc2, int* m, int* M, int i, int j, int width, int height)
{
	float eps = 0.1;

	*m = 0;
	*M = 0;

	if ((sqrLength(desc1.minutias[i], desc2.minutias[j]) < COMPARE_RADIUS*COMPARE_RADIUS)
		&& ((desc1.minutias[i].angle - desc2.minutias[j].angle) < eps))
	{
		*m = 1;
		*M = 1;
	}
	else
	{
		if ((sqrLength(desc1.minutias[i], desc2.center) < FENG_CONSTANT * DESCRIPTOR_RADIUS * DESCRIPTOR_RADIUS) &&
			(desc1.minutias[i].x >= 0 && desc1.minutias[i].x < width
			&& desc1.minutias[i].y >= 0 && desc1.minutias[i].y < height))
		{
			*M = 1;
		}
	}
}

__global__ void compareDescriptors(Descriptor* input, Descriptor** current, Descriptor** temp0, Descriptor** temp1, 
	float*** s, int height, int width) //block 16*16*2
{ 
	__shared__ int cache_m[DESC_BLOCK_SIZE][DESC_BLOCK_SIZE][2];
	__shared__ int cache_M[DESC_BLOCK_SIZE][DESC_BLOCK_SIZE][2];
	
	int row = defaultRow();
	int column = defaultColumn();
	int x = defaultDescriptorRow();
	int y = defaultDescriptorColumn();
	int k = defaultFinger();
	
	int cacheIdxX = threadIdx.y;
	int cacheIdxY = threadIdx.x;
	int cacheIdxZ = threadIdx.z;
	
	float eps = 0.1f;

	if ((cacheIdxX == 0) && (cacheIdxZ == 0))
	{
		transformate(input[x].minutias, current[k][y].center, temp0[k][y].minutias, column);
	}
	else if ((cacheIdxX == 0) && (cacheIdxZ == 1))
	{
		transformate(current[k][y].minutias, input[x].center, temp1[k][y].minutias, column);
	}
	__syncthreads();
		
	if (cacheIdxZ == 0)
	{
		matchingPoints(temp0[k][y], current[k][y], &cache_m[cacheIdxX][cacheIdxY][0],
			&cache_M[cacheIdxX][cacheIdxY][0], row, column, width, height);
	}
	else
	{
		matchingPoints(temp1[k][y], input[x], &cache_m[cacheIdxX][cacheIdxY][1],
			&cache_M[cacheIdxX][cacheIdxY][1], row, column, width, height);
	}

	__syncthreads();

	int i = DESC_BLOCK_SIZE / 2;
	while (i != 0)
	{
		if (cacheIdxX < i)
		{
			cache_m[cacheIdxX][cacheIdxY][cacheIdxZ] += cache_m[cacheIdxX + i][cacheIdxY][cacheIdxZ];
		}
		else 
		{
			cache_M[cacheIdxX - i][cacheIdxY][cacheIdxZ] += cache_m[cacheIdxX][cacheIdxY][cacheIdxZ];
		}

		__syncthreads();
		i /= 2;
	}

	i = DESC_BLOCK_SIZE / 2;
	while (i != 0)
	{
		if (cacheIdxX == 0)
		{
			if (cacheIdxY < i)
			{
				cache_m[cacheIdxX][cacheIdxY][cacheIdxZ] += cache_m[cacheIdxX][cacheIdxY + i][cacheIdxZ];
			}
			else
			{
				cache_M[cacheIdxX][cacheIdxY][cacheIdxZ] += cache_m[cacheIdxX][cacheIdxY - i][cacheIdxZ];
			}
		}

		__syncthreads();
		i /= 2;
	}

	if ((cacheIdxX == 0) && (cacheIdxY == 0) && (cacheIdxZ == 0))
	{
		s[k][blockIdx.y][blockIdx.x] = (cache_m[0][0][0] + 1.0f)*(cache_m[0][0][1] + 1.0f) / (cache_M[0][0][0] + 1.0f) / (cache_M[0][0][1] + 1.0f);
	}
}

#ifdef DEBUG
int main()
{

	return 0;
}
#endif*/